#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include "test_kernels.cuh"
#include <iostream>

template <class T>
int device_populate_random(T* d_elements, unsigned n) {
    T h_elements[n];
    for (unsigned i = 0; i < n; i++)
        h_elements[i] = T::rand_host();
    return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

template <class T>
int device_set(T* d_elements, T el, unsigned n) {
    T h_elements[n];
    for (unsigned i = 0; i < n; i++)
        h_elements[i] = el;
    return hipMemcpy(d_elements, h_elements, sizeof(T) * n, hipMemcpyHostToDevice);
}

class PrimitivesTest : public ::testing::Test {
protected:
  static const unsigned n = 1 << 5;

  proj *points1{};
  proj *points2{};
  scalar_field *scalars1{};
  scalar_field *scalars2{};
  proj *zero_points{};
  scalar_field *zero_scalars{};
  scalar_field *one_scalars{};
  affine *aff_points{};
  proj *res_points1{};
  proj *res_points2{};
  scalar_field *res_scalars1{};
  scalar_field *res_scalars2{};
  scalar_field::wide *res_scalars_wide{};
  scalar_field::wide *res_scalars_wide_full{};

  PrimitivesTest() {
    assert(!hipDeviceReset());
    assert(!hipMallocManaged(&points1, n * sizeof(proj)));
    assert(!hipMallocManaged(&points2, n * sizeof(proj)));
    assert(!hipMallocManaged(&scalars1, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&scalars2, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&zero_points, n * sizeof(proj)));
    assert(!hipMallocManaged(&zero_scalars, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&one_scalars, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&aff_points, n * sizeof(affine)));
    assert(!hipMallocManaged(&res_points1, n * sizeof(proj)));
    assert(!hipMallocManaged(&res_points2, n * sizeof(proj)));
    assert(!hipMallocManaged(&res_scalars1, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&res_scalars2, n * sizeof(scalar_field)));
    assert(!hipMallocManaged(&res_scalars_wide, n * sizeof(scalar_field::wide)));
    assert(!hipMallocManaged(&res_scalars_wide_full, n * sizeof(scalar_field::wide)));

  }

  ~PrimitivesTest() override {
    hipFree(points1);
    hipFree(points2);
    hipFree(scalars1);
    hipFree(scalars2);
    hipFree(zero_points);
    hipFree(zero_scalars);
    hipFree(one_scalars);
    hipFree(aff_points);
    hipFree(res_points1);
    hipFree(res_points2);
    hipFree(res_scalars1);
    hipFree(res_scalars2);

    hipFree(res_scalars_wide);
    hipFree(res_scalars_wide_full);

    hipDeviceReset();
  }

  void SetUp() override {
    ASSERT_EQ(device_populate_random<proj>(points1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<proj>(points2, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_field>(scalars1, n), hipSuccess);
    ASSERT_EQ(device_populate_random<scalar_field>(scalars2, n), hipSuccess);
    ASSERT_EQ(device_set<proj>(zero_points, proj::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<scalar_field>(zero_scalars, scalar_field::zero(), n), hipSuccess);
    ASSERT_EQ(device_set<scalar_field>(one_scalars, scalar_field::one(), n), hipSuccess);
    ASSERT_EQ(hipMemset(aff_points, 0, n * sizeof(affine)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points1, 0, n * sizeof(proj)), hipSuccess);
    ASSERT_EQ(hipMemset(res_points2, 0, n * sizeof(proj)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars1, 0, n * sizeof(scalar_field)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars2, 0, n * sizeof(scalar_field)), hipSuccess);
    
    ASSERT_EQ(hipMemset(res_scalars_wide, 0, n * sizeof(scalar_field::wide)), hipSuccess);
    ASSERT_EQ(hipMemset(res_scalars_wide_full, 0, n * sizeof(scalar_field::wide)), hipSuccess);
  }
};

TEST_F(PrimitivesTest, FieldAdditionSubtractionCancel) {
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_scalars1, scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars2[i]);
}

TEST_F(PrimitivesTest, FieldZeroAddition) {
  ASSERT_EQ(vec_add(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldAdditionHostDeviceEq) {
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] + scalars2[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByOne) {
  ASSERT_EQ(vec_mul(scalars1, one_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByMinusOne) {
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, res_scalars1, res_scalars2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, res_scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], zero_scalars[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByZero) {
  ASSERT_EQ(vec_mul(scalars1, zero_scalars, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(zero_scalars[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationInverseCancel) {
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars2, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i], res_scalars1[i] * res_scalars2[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationHostDeviceEq) {
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars2[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationByTwoEqSum) {
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars2[i], scalars1[i] + scalars1[i]);
}

TEST_F(PrimitivesTest, FieldSqrHostDeviceEq) {
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * scalars1[i], res_scalars1[i]);
}

TEST_F(PrimitivesTest, FieldMultiplicationSqrEq) {
  ASSERT_EQ(vec_mul(scalars1, scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(field_vec_sqr(scalars1, res_scalars2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i], res_scalars2[i]);
}

TEST_F(PrimitivesTest, ECRandomPointsAreOnCurve) {
  for (unsigned i = 0; i < n; i++)
    ASSERT_PRED1(proj::is_on_curve, points1[i]);
}

TEST_F(PrimitivesTest, ECPointAdditionSubtractionCancel) {
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_sub(res_points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECPointZeroAddition) {
  ASSERT_EQ(vec_add(points1, zero_points, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECPointAdditionHostDeviceEq) {
  ASSERT_EQ(vec_add(points1, points2, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i] + points2[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationHostDeviceEq) {
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(scalars1[i] * points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByOne) {
  ASSERT_EQ(vec_mul(one_scalars, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByMinusOne) {
  ASSERT_EQ(vec_neg(one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationByTwo) {
  ASSERT_EQ(vec_add(one_scalars, one_scalars, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ((one_scalars[i] + one_scalars[i]) * points1[i], res_points1[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationInverseCancel) {
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(field_vec_inv(scalars1, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, res_points1, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationIsDistributiveOverMultiplication) {
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, res_points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars1, scalars2, res_scalars1, n), hipSuccess);
  ASSERT_EQ(vec_mul(res_scalars1, points1, res_points1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECScalarMultiplicationIsDistributiveOverAddition) {
  ASSERT_EQ(vec_mul(scalars1, points1, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_mul(scalars2, points1, res_points2, n), hipSuccess);
  ASSERT_EQ(vec_add(scalars1, scalars2, res_scalars1, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_scalars1[i] * points1[i], res_points1[i] + res_points2[i]);
}

TEST_F(PrimitivesTest, ECProjectiveToAffine) {
  ASSERT_EQ(point_vec_to_affine(points1, aff_points, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(points1[i], proj::from_affine(aff_points[i]));
}

TEST_F(PrimitivesTest, ECMixedPointAddition) {
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_add(points1, points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], res_points2[i]);
}

TEST_F(PrimitivesTest, ECMixedAdditionOfNegatedPointEqSubtraction) {
  ASSERT_EQ(point_vec_to_affine(points2, aff_points, n), hipSuccess);
  ASSERT_EQ(vec_sub(points1, aff_points, res_points1, n), hipSuccess);
  ASSERT_EQ(vec_neg(points2, res_points2, n), hipSuccess);
  for (unsigned i = 0; i < n; i++)
    ASSERT_EQ(res_points1[i], points1[i] + res_points2[i]);
}

TEST_F(PrimitivesTest, MP_LSB_MULT) {


  ASSERT_EQ(mp_lsb_mult(scalars1, scalars2, res_scalars_wide), hipSuccess);
  std::cout << "GPU lsb mult output  = 0x";
  for (int i=0; i<2*scalar_field::TLC; i++)
  {
    std::cout << std::hex << res_scalars_wide[0].limbs_storage.limbs[i];
  }
  std::cout << std::endl;


  ASSERT_EQ(mp_mult(scalars1, scalars2, res_scalars_wide_full), hipSuccess);
  std::cout << "GPU full mult output = 0x";
  for (int i=0; i<2*scalar_field::TLC; i++)
  {
    std::cout << std::hex << res_scalars_wide_full[0].limbs_storage.limbs[i];
  }
  std::cout << std::endl;

  for (int i=0; i<2*scalar_field::TLC; i++)
  {
    if (res_scalars_wide_full[0].limbs_storage.limbs[i] == res_scalars_wide[0].limbs_storage.limbs[i])
    std::cout << "matched index = " << i << std::endl;
  }

}


int main(int argc, char **argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
