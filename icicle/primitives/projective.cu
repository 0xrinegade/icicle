#include <hip/hip_runtime.h>
#include "../curves/curve_config.cuh"
#include "projective.cuh"

extern "C" bool eq(projective_t *point1, projective_t *point2, size_t device_id = 0)
{
  return (*point1 == *point2);
}

#if defined(G2_DEFINED)
extern "C" bool eq_g2(g2_projective_t *point1, g2_projective_t *point2, size_t device_id = 0)
{
  return (*point1 == *point2);
}
#endif