
#include <hip/hip_runtime.h>

template <class T>
__global__ void add_elements_kernel(const T* x, const T* y, T* result, const unsigned count)
{
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= count) return;
  result[tid] = x[tid] + y[tid];
}

template <class T>
__global__ void fake_ntt_kernel(const T* x, T* result, const unsigned thread_count)
{
  extern __shared__ T shmem[];
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= thread_count) return;
  shmem[4*threadIdx.x] = x[4*tid] + x[4*tid+1];
  shmem[4*threadIdx.x+1] = x[4*tid] + T::neg(x[4*tid+1]);
  shmem[4*threadIdx.x+2] = x[4*tid+2] + x[4*tid+3];
  shmem[4*threadIdx.x+3] = x[4*tid+2] + T::neg(x[4*tid+3]);
  __syncthreads();
  result[4*tid] = shmem[2*threadIdx.x] + shmem[2*threadIdx.x + 4*blockDim.x/2];
  result[4*tid+1] = shmem[2*threadIdx.x] + T::neg(shmem[2*threadIdx.x + 4*blockDim.x/2]);
  result[4*tid+2] = shmem[2*threadIdx.x+1] + shmem[2*threadIdx.x + 4*blockDim.x/2+1];
  result[4*tid+3] = shmem[2*threadIdx.x+1] + T::neg(shmem[2*threadIdx.x + 4*blockDim.x/2+1]);
}


template <class T>
__global__ void bugged_add_elements_kernel(const T* x, const T* y, T* result, const unsigned count)
{
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  // if (tid >= count) return;
  // printf("tid %d\n", tid);
  result[tid] = x[tid] + y[tid];
}

template <class T>
__global__ void bugged_fake_ntt_kernel(const T* x, T* result, const unsigned thread_count)
{
  extern __shared__ T shmem[];
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  // if (tid >= thread_count) return;
  // if (tid == 0){
  //   for (int i = 0; i < 8; i++)
  //   {
  //     shmem[i]=T::zero();
  //   }
  // }

  shmem[4*threadIdx.x] = x[4*tid] + x[4*tid+1];
  shmem[4*threadIdx.x+1] = x[4*tid] + T::neg(x[4*tid+1]);
  shmem[4*threadIdx.x+2] = x[4*tid+2] + x[4*tid+1];
  shmem[4*threadIdx.x+4] = x[4*tid+2] + T::neg(x[4*tid+1]);

  __syncthreads();

  // if (tid == 0){
  //   for (int i = 0; i < 8; i++)
  //   {
  //     printf("%d ",shmem[i]);
  //   }
  //   printf("\n");
  // }

  // printf("tid: %d, addr1: %d, addr2: %d\n", tid, 2*threadIdx.x, 2*threadIdx.x + 4*blockDim.x);
  result[4*tid] = shmem[2*threadIdx.x] + shmem[2*threadIdx.x + 4*blockDim.x];  // Incorrect offset
  result[4*tid+1] = shmem[2*threadIdx.x] + T::neg(shmem[2*threadIdx.x + 4*blockDim.x]);  // Incorrect offset
  result[4*tid+2] = shmem[2*threadIdx.x+1] + shmem[2*threadIdx.x + 4*blockDim.x+1];  // Incorrect offset
  result[4*tid+3] = shmem[2*threadIdx.x+1] + T::neg(shmem[2*threadIdx.x +4*blockDim.x+1]);  // Incorrect offset
}

template <class T>
__global__ void bucket_acc_naive(T* buckets, unsigned* indices, unsigned* sizes, unsigned nof_buckets){
  int tid = blockDim.x*blockIdx.x + threadIdx.x;
  if (tid >= nof_buckets) return;
  for (int i = 0; i < sizes[tid]; i++)
  {
    buckets[indices[tid]] = buckets[indices[tid]] + buckets[indices[tid]];
  }
}

template <class T>
__global__ void bucket_acc_reg(T* buckets, unsigned* indices, unsigned* sizes, unsigned nof_buckets){
  int tid = blockDim.x*blockIdx.x + threadIdx.x;
  if (tid >= nof_buckets) return;
  T bucket = buckets[indices[tid]];
  for (int i = 0; i < sizes[tid]; i++)
  {
    bucket = bucket + bucket;
  }
  buckets[indices[tid]] = bucket;
}

template <class T>
__global__ void bucket_acc_memory_baseline(T* buckets1, T* buckets2, unsigned* indices, unsigned nof_buckets){
  int tid = blockDim.x*blockIdx.x + threadIdx.x;
  if (tid >= nof_buckets) return;
  buckets2[indices[tid]] = buckets1[indices[tid]];
}

template <class T>
__global__ void simple_memory_copy(T* buckets1, T* buckets2, unsigned nof_buckets){
  int tid = blockDim.x*blockIdx.x + threadIdx.x;
  if (tid >= nof_buckets) return;
  buckets2[tid] = buckets1[tid];
}

template <class T>
__global__ void bucket_acc_compute_baseline(T* buckets, unsigned* indices, unsigned* sizes, unsigned nof_buckets){
  int tid = blockDim.x*blockIdx.x + threadIdx.x;
  if (tid >= nof_buckets) return;
  T bucket = buckets[indices[tid]];
  for (int j = 0; j < 100; j++)
  {
    for (int i = 0; i < sizes[tid]; i++)
    {
      bucket = bucket + bucket;
    }
  }
  buckets[indices[tid]] = bucket;
}