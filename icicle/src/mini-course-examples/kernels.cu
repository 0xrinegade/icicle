
#include <hip/hip_runtime.h>

template <class T>
__global__ void add_elements_kernel(const T* x, const T* y, T* result, const unsigned count)
{
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= count) return;
  // printf("tid %d\n", tid);
  result[tid] = x[tid] + y[tid];
}

template <class T>
__global__ void fake_ntt_kernel(const T* x, T* result, const unsigned thread_count)
{
  extern __shared__ T shmem[];
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= thread_count) return;
  shmem[4*threadIdx.x] = x[4*tid] + x[4*tid+1];
  shmem[4*threadIdx.x+1] = x[4*tid] + T::neg(x[4*tid+1]);
  shmem[4*threadIdx.x+2] = x[4*tid+2] + x[4*tid+3];
  shmem[4*threadIdx.x+3] = x[4*tid+2] + T::neg(x[4*tid+3]);
  __syncthreads();
  result[4*tid] = shmem[2*threadIdx.x] + shmem[2*threadIdx.x + 4*blockDim.x/2];
  result[4*tid+1] = shmem[2*threadIdx.x] + T::neg(shmem[2*threadIdx.x + 4*blockDim.x/2]);
  result[4*tid+2] = shmem[2*threadIdx.x+1] + shmem[2*threadIdx.x + 4*blockDim.x/2+1];
  result[4*tid+3] = shmem[2*threadIdx.x+1] + T::neg(shmem[2*threadIdx.x + 4*blockDim.x/2+1]);
}

template <class T>
__global__ void bugged_fake_ntt_kernel(const T* x, T* result, const unsigned thread_count)
{
  extern __shared__ T shmem[];
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  // if (tid >= thread_count) return;
  // if (tid == 0){
  //   for (int i = 0; i < 8; i++)
  //   {
  //     shmem[i]=T::zero();
  //   }
  // }

  shmem[4*threadIdx.x] = x[4*tid] + x[4*tid+1];
  shmem[4*threadIdx.x+1] = x[4*tid] + T::neg(x[4*tid+1]);
  shmem[4*threadIdx.x+2] = x[4*tid+2] + x[4*tid+1];
  shmem[4*threadIdx.x+4] = x[4*tid+2] + T::neg(x[4*tid+1]);

  __syncthreads();

  // if (tid == 0){
  //   for (int i = 0; i < 8; i++)
  //   {
  //     printf("%d ",shmem[i]);
  //   }
  //   printf("\n");
  // }

  // printf("tid: %d, addr1: %d, addr2: %d\n", tid, 2*threadIdx.x, 2*threadIdx.x + 4*blockDim.x);
  result[4*tid] = shmem[2*threadIdx.x] + shmem[2*threadIdx.x + 4*blockDim.x];  // Incorrect offset
  result[4*tid+1] = shmem[2*threadIdx.x] + T::neg(shmem[2*threadIdx.x + 4*blockDim.x]);  // Incorrect offset
  result[4*tid+2] = shmem[2*threadIdx.x+1] + shmem[2*threadIdx.x + 4*blockDim.x+1];  // Incorrect offset
  result[4*tid+3] = shmem[2*threadIdx.x+1] + T::neg(shmem[2*threadIdx.x +4*blockDim.x+1]);  // Incorrect offset
}