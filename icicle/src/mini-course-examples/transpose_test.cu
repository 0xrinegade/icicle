#include "hip/hip_runtime.h"
#include "fields/id.h"
#define FIELD_ID 1001
// #define CURVE_ID 3
// #include "curves/curve_config.cuh"
#include "fields/field_config.cuh"

#include <chrono>
#include <iostream>
#include <vector>
#include <random>
#include <cub/device/device_radix_sort.cuh>

#include "fields/field.cuh"
#include "curves/projective.cuh"
#include "gpu-utils/device_context.cuh"

#include "kernels.cu"

class Dummy_Scalar
{
public:
  static constexpr unsigned NBITS = 32;

  unsigned x;
  unsigned p = 10;
  // unsigned p = 1<<30;

  static HOST_DEVICE_INLINE Dummy_Scalar zero() { return {0}; }

  static HOST_DEVICE_INLINE Dummy_Scalar one() { return {1}; }

  friend HOST_INLINE std::ostream& operator<<(std::ostream& os, const Dummy_Scalar& scalar)
  {
    os << scalar.x;
    return os;
  }

  HOST_DEVICE_INLINE unsigned get_scalar_digit(unsigned digit_num, unsigned digit_width) const
  {
    return (x >> (digit_num * digit_width)) & ((1 << digit_width) - 1);
  }

  friend HOST_DEVICE_INLINE Dummy_Scalar operator+(Dummy_Scalar p1, const Dummy_Scalar& p2)
  {
    return {(p1.x + p2.x) % p1.p};
  }

  friend HOST_DEVICE_INLINE bool operator==(const Dummy_Scalar& p1, const Dummy_Scalar& p2) { return (p1.x == p2.x); }

  friend HOST_DEVICE_INLINE bool operator==(const Dummy_Scalar& p1, const unsigned p2) { return (p1.x == p2); }

  static HOST_DEVICE_INLINE Dummy_Scalar neg(const Dummy_Scalar& scalar) { return {scalar.p - scalar.x}; }
  static HOST_INLINE Dummy_Scalar rand_host()
  {
    return {(unsigned)rand() % 10};
    // return {(unsigned)rand()};
  }
};


typedef field_config::scalar_t test_scalar;
// typedef curve_config::scalar_t test_scalar;
// typedef curve_config::projective_t test_projective;
// typedef curve_config::affine_t test_affine;

// typedef int test_t;
// typedef int4 test_t;
// typedef Dummy_Scalar test_t;
// typedef test_projective test_t;
typedef test_scalar test_t;

int main()
{

  hipEvent_t start, stop;
  float kernel_time;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  int N = 1<<11;
  int N2 = N*N;
  
  test_t* arr1_h = new test_t[N2];
  test_t* arr2_h = new test_t[N2];

  test_t *arr1_d, *arr2_d;
  
  hipMalloc(&arr1_d, N2*sizeof(test_t));
  hipMalloc(&arr2_d, N2*sizeof(test_t));

  for (int i = 0; i < N2; i++)
  {
    arr1_h[i] = i > 100? arr1_h[i-100] : test_t::rand_host();
  }
  
  hipMemcpy(arr1_d, arr1_h, sizeof(test_t) * N2, hipMemcpyHostToDevice);

  int THREADS = 256;
  int BLOCKS = (N2 + THREADS - 1)/THREADS;
  
  //warm up
  simple_memory_copy<<<BLOCKS, THREADS>>>(arr1_d, arr2_d, N2);
  shmem_transpose<<<BLOCKS, THREADS>>>(arr1_d, arr2_d, N);
  hipDeviceSynchronize();
  std::cout << "cuda err: " << hipGetErrorString(hipGetLastError()) << std::endl;

  hipEventRecord(start, 0);

  simple_memory_copy<<<BLOCKS, THREADS>>>(arr1_d, arr2_d, N2);
  // naive_transpose_write<<<BLOCKS, THREADS>>>(arr1_d, arr2_d, N);
  // naive_transpose_read<<<BLOCKS, THREADS>>>(arr1_d, arr2_d, N);
  // shmem_transpose<<<BLOCKS, THREADS>>>(arr1_d, arr2_d, N);
  
  hipDeviceSynchronize();
  std::cout << "cuda err: " << hipGetErrorString(hipGetLastError()) << std::endl;
  hipEventRecord(stop, 0);
  hipStreamSynchronize(0);
  hipEventElapsedTime(&kernel_time, start, stop);
  printf("kernel_time : %.3f ms.\n", kernel_time);

  return 0;
}