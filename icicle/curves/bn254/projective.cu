#include <hip/hip_runtime.h>
#include "c_api.h"
#include "curve_config.cuh"
#include "../../primitives/projective.cuh"

extern "C" BN254_projective_t* create_projective() {
    return reinterpret_cast<BN254_projective_t*>(new BN254::projective_t);
}

extern "C" void delete_projective(BN254_projective_t* p) {
    delete reinterpret_cast<BN254::projective_t*>(p);
}

extern "C" bool eq_bn254(BN254::projective_t *point1, BN254::projective_t *point2, size_t device_id = 0)
{
    return (*point1 == *point2);
}