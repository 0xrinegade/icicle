#include "hip/hip_runtime.h"
#ifndef MSM
#define MSM
#pragma once
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "../../primitives/affine.cuh"
#include <iostream>
#include <vector>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_scan.cuh>
#include "../../utils/cuda_utils.cuh"
#include "../../primitives/projective.cuh"
#include "../../primitives/field.cuh"
#include "msm.cuh"


#define BIG_TRIANGLE
// #define SSM_SUM  //WIP

template <typename T>
static constexpr __device__ __forceinline__ T ld_single(const T *ptr) {
return __ldg(ptr);
};

template <class T, typename U, unsigned STRIDE>
static constexpr __device__ __forceinline__ T ld(const T *address, const unsigned offset) {
  static_assert(alignof(T) % alignof(U) == 0);
  static_assert(sizeof(T) % sizeof(U) == 0);
  constexpr size_t count = sizeof(T) / sizeof(U);
  T result = {};
  auto pa = reinterpret_cast<const U *>(address) + offset;
  auto pr = reinterpret_cast<U *>(&result);
#pragma unroll
  for (unsigned i = 0; i < count; i++) {
    const auto pai = pa + i * STRIDE;
    const auto pri = pr + i;
    *pri = ld_single<U>(pai);
  }
  return result;
}

template <class T, unsigned STRIDE = 1, typename U = std::enable_if_t<sizeof(T) % sizeof(uint4) == 0, uint4>>
static constexpr __device__ __forceinline__ T memory_load(const T *address, const unsigned offset = 0, [[maybe_unused]] uint4 _dummy = {}) {
  return ld<T, U, STRIDE>(address, offset);
};

template <class T, unsigned STRIDE = 1, typename U = std::enable_if_t<(sizeof(T) % sizeof(uint4) != 0) && (sizeof(T) % sizeof(uint2) == 0), uint2>>
static constexpr __device__ __forceinline__ T memory_load(const T *address, const unsigned offset = 0, [[maybe_unused]] uint2 _dummy = {}) {
  return ld<T, U, STRIDE>(address, offset);
};

template <class T, unsigned STRIDE = 1, typename U = std::enable_if_t<sizeof(T) % sizeof(uint2) != 0, unsigned>>
static constexpr __device__ __forceinline__ T memory_load(const T *address, const unsigned offset = 0, [[maybe_unused]] unsigned _dummy = {}) {
  return ld<T, U, STRIDE>(address, offset);
};

//this kernel performs single scalar multiplication
//each thread multilies a single scalar and point
template <typename P, typename S>
__global__ void ssm_kernel(S *scalars, P *points, P *results, unsigned N) {

  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) results[tid] = scalars[tid]*points[tid];

}

//this kernel sums all the elements in a given vector using multiple threads
template <typename P>
__global__ void sum_reduction_kernel(P *v, P* v_r) {

	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Start at 1/2 block stride and divide by two each iteration
	for (unsigned s = blockDim.x / 2; s > 0; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			v[tid] = v[tid] + v[tid + s];
		}
    __syncthreads();
	}

	// Let the thread 0 for this block write the final result
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = v[tid];
	}
}

//this kernel initializes the buckets with zero points
//each thread initializes a different bucket
template <typename P>
__global__ void initialize_buckets_kernel(P *buckets, unsigned N) {
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) buckets[tid] = P::zero(); //zero point

}

//this kernel splits the scalars into digits of size c
//each thread splits a single scalar into nof_bms digits
template <typename S>
__global__ void split_scalars_kernel(unsigned *buckets_indices, unsigned *point_indices, S *scalars, unsigned total_size, unsigned msm_log_size, unsigned nof_bms, unsigned bm_bitsize, unsigned c, unsigned top_bm_nof_missing_bits){
  
  constexpr unsigned sign_mask = 0x80000000;
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned bucket_index;
  unsigned bucket_index2;
  unsigned current_index;
  unsigned msm_index = tid >> msm_log_size;
  unsigned borrow = 0;
  if (tid < total_size){
    S scalar = scalars[tid];
    if (tid == 0) printf("scalar %u", scalar);

    for (unsigned bm = 0; bm < nof_bms; bm++)
    {
      // bucket_index = scalar.get_scalar_digit(bm, c) + (bm==nof_bms-1? ((tid&top_bm_nof_missing_bits)<<(c-top_bm_nof_missing_bits)) : 0);
      bucket_index = scalar.get_scalar_digit(bm, c);
      bucket_index += borrow;
      borrow = 0;
      unsigned sign = 0;
      if (tid == 0) printf("index %u", bucket_index);
      if (bucket_index > (1<<(c-1))) {
        bucket_index = (1 << c) - bucket_index;
        borrow = 1;
        sign = sign_mask;
      }
      if (tid == 0) printf("new index %u", bucket_index);
      // if (bm==nof_bms-1) {
      //   bucket_index2 = bucket_index + ((tid&((1<<top_bm_nof_missing_bits)-1))<<(c-top_bm_nof_missing_bits));
      //   if (tid<10) printf("tid %u bi1 %u bi2 %u\n",tid, bucket_index, bucket_index2);
      //   bucket_index = bucket_index2;
      // }
      current_index = bm * total_size + tid;
      buckets_indices[current_index] = (msm_index<<(c+bm_bitsize)) | (bm<<c) | bucket_index;  //the bucket module number and the msm number are appended at the msbs
      // buckets_indices[current_index] = (msm_index<<(c-1+bm_bitsize)) | (bm<<(c-1)) | bucket_index;  //the bucket module number and the msm number are appended at the msbs
      // point_indices[current_index] = tid; //the point index is saved for later
      point_indices[current_index] = sign | tid; //the point index is saved for later
    }
  }
}

//this kernel adds up the points in each bucket
// __global__ void accumulate_buckets_kernel(P *__restrict__ buckets, unsigned *__restrict__ bucket_offsets,
  //  unsigned *__restrict__ bucket_sizes, unsigned *__restrict__ single_bucket_indices, unsigned *__restrict__ point_indices, A *__restrict__ points, unsigned nof_buckets, unsigned batch_size, unsigned msm_idx_shift){
template <typename P, typename A>
__global__ void accumulate_buckets_kernel(P *__restrict__ buckets, const unsigned *__restrict__ bucket_offsets, const unsigned *__restrict__ bucket_sizes, const unsigned *__restrict__ single_bucket_indices, const unsigned *__restrict__ point_indices, A *__restrict__ points, const unsigned nof_buckets, const unsigned *nof_buckets_to_compute, const unsigned msm_idx_shift, const unsigned c){
  
  constexpr unsigned sign_mask = 0x80000000;
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  // if (tid>=*nof_buckets_to_compute || tid<11){ 
  if (tid>=*nof_buckets_to_compute){ 
    return;
  }
  const unsigned msm_index = single_bucket_indices[tid]>>msm_idx_shift;
  const unsigned bm_index = (single_bucket_indices[tid]&((1<<msm_idx_shift)-1))>>c;
  const unsigned bucket_index = msm_index * nof_buckets + bm_index * ((1<<(c-1))+1) + (single_bucket_indices[tid]&((1<<c)-1));
  const unsigned bucket_offset = bucket_offsets[tid];
  // if (tid<10) printf("tid %u size %u\n", tid, bucket_sizes[tid]);
  // if (tid==0) return;
  // if ((bucket_index>>20)==13) return;
  // if (bucket_sizes[tid]==16777216) printf("tid %u size %u bucket %u offset %u\n", tid, bucket_sizes[tid], bucket_index, bucket_offset);
  // const unsigned *indexes = point_indices + bucket_offset;
  P bucket = P::zero(); //todo: get rid of init buckets? no.. because what about buckets with no points
  // unsigned point_ind;
  for (unsigned i = 0; i < bucket_sizes[tid]; i++)  //add the relevant points starting from the relevant offset up to the bucket size
  {
    // unsigned point_ind = *indexes++;
    // auto point = memory_load<A>(points + point_ind);
    // point_ind = point_indices[bucket_offset+i];
    // bucket = bucket + P::one();
    unsigned point_ind = point_indices[bucket_offset+i];
    unsigned sign = point_ind & sign_mask;
    point_ind &= ~sign_mask;
    // printf("tid %u sign %u point ind %u \n", tid,sign, point_ind);
    A point = points[point_ind];
    if (sign) point = A::neg(point);
    bucket = bucket + point;
    // const unsigned* pa = reinterpret_cast<const unsigned*>(points[point_ind]);
    // P point;
    // Dummy_Scalar scal;
    // scal.x = __ldg(pa);
    // point.x = scal;
    // bucket = bucket + point;
  }
  // buckets[tid] = bucket;
  buckets[bucket_index] = bucket;
}

//this kernel sums the entire bucket module
//each thread deals with a single bucket module
template <typename P>
__global__ void big_triangle_sum_kernel(P* buckets, P* final_sums, unsigned nof_bms, unsigned c){

  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid>=nof_bms) return;
  P line_sum = buckets[(tid+1)*((1<<c)+1)-1];
  final_sums[tid] = line_sum;
  for (unsigned i = (1<<c)-1; i >0; i--)
  {
    line_sum = line_sum + buckets[tid*((1<<c)+1) + i];  //using the running sum method
    final_sums[tid] = final_sums[tid] + line_sum;
  }
}

//this kernel uses single scalar multiplication to multiply each bucket by its index
//each thread deals with a single bucket
template <typename P, typename S>
__global__ void ssm_buckets_kernel(P* buckets, unsigned* single_bucket_indices, unsigned nof_buckets, unsigned c){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid>nof_buckets) return;
  unsigned bucket_index = single_bucket_indices[tid];
  S scalar_bucket_multiplier;
  scalar_bucket_multiplier = {bucket_index&((1<<c)-1), 0, 0, 0, 0, 0, 0, 0}; //the index without the bucket module index
  buckets[bucket_index] = scalar_bucket_multiplier*buckets[bucket_index];

}

//this kernel computes the final result using the double and add algorithm
//it is done by a single thread
template <typename P, typename S>
__global__ void final_accumulation_kernel(P* final_sums, P* final_results, unsigned nof_msms, unsigned nof_bms, unsigned c){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid>nof_msms) return;
  P final_result = P::zero();
  for (unsigned i = nof_bms; i >1; i--)
  {
    final_result = final_result + final_sums[i-1 + tid*nof_bms];  //add
    for (unsigned j=0; j<c; j++)  //double
    {
      final_result = final_result + final_result;
    }
  }
  final_results[tid] = final_result + final_sums[tid*nof_bms];

}

//this function computes msm using the bucket method
template <typename S, typename P, typename A>
void bucket_method_msm(unsigned bitsize, unsigned c, S *scalars, A *points, unsigned size, P* final_result, bool on_device) {
  
  // std::cout<<"points"<<std::endl;
  // for (int i = 0; i < size; i++)
  // {
  //   std::cout<<points[i]<<" ";
  // }
  // std::cout<<std::endl;
  // std::cout<<"scalars"<<std::endl;
  // for (int i = 0; i < size; i++)
  // {
  //   std::cout<<scalars[i]<<" ";
  // }
  // std::cout<<std::endl;

  S *d_scalars;
  A *d_points;
  if (!on_device) {
    //copy scalars and point to gpu
    hipMalloc(&d_scalars, sizeof(S) * size);
    hipMalloc(&d_points, sizeof(A) * size);
    hipMemcpy(d_scalars, scalars, sizeof(S) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_points, points, sizeof(A) * size, hipMemcpyHostToDevice);
  }
  else {
    d_scalars = scalars;
    d_points = points;
  }

  P *buckets;
  //compute number of bucket modules and number of buckets in each module
  unsigned nof_bms = bitsize/c;
  unsigned msm_log_size = ceil(log2(size));
  unsigned bm_bitsize = ceil(log2(nof_bms));
  if (bitsize%c){
    nof_bms++;
  }
  unsigned top_bm_nof_missing_bits = c*nof_bms - bitsize;
  std::cout << "top_bm_nof_missing_bits" << top_bm_nof_missing_bits <<std::endl;
  // unsigned nof_buckets = nof_bms<<c;
  unsigned nof_buckets = nof_bms*((1<<(c-1))+1); //signed digits
  hipMalloc(&buckets, sizeof(P) * nof_buckets);

  // launch the bucket initialization kernel with maximum threads
  unsigned NUM_THREADS = 1 << 10;
  unsigned NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
  initialize_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, nof_buckets);
  hipDeviceSynchronize();
  printf("cuda error %u",hipGetLastError());

  unsigned *bucket_indices;
  unsigned *point_indices;
  hipMalloc(&bucket_indices, sizeof(unsigned) * size * (nof_bms+1));
  hipMalloc(&point_indices, sizeof(unsigned) * size * (nof_bms+1));

  //split scalars into digits
  NUM_THREADS = 1 << 10;
  NUM_BLOCKS = (size * (nof_bms+1) + NUM_THREADS - 1) / NUM_THREADS;
  split_scalars_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(bucket_indices + size, point_indices + size, d_scalars, size, msm_log_size, 
                                                    nof_bms, bm_bitsize, c, top_bm_nof_missing_bits); //+size - leaving the first bm free for the out of place sort later
                                                    hipDeviceSynchronize();
                                                    printf("cuda error %u",hipGetLastError());


  // hipDeviceSynchronize();
  // std::vector<unsigned> h_bucket_ind;
  // std::vector<unsigned> h_point_ind;
  // h_bucket_ind.reserve(size * (nof_bms+1));
  // h_point_ind.reserve(size * (nof_bms+1));
  // hipMemcpy(h_bucket_ind.data(), bucket_indices, sizeof(unsigned) * size * (nof_bms+1), hipMemcpyDeviceToHost);
  // hipMemcpy(h_point_ind.data(), point_indices, sizeof(unsigned) * size * (nof_bms+1), hipMemcpyDeviceToHost);
  //   std::cout<<hipGetLastError()<<std::endl;
  // std::cout<<"buckets inds"<<std::endl;
  // for (int i = 0; i < size * (nof_bms+1); i++)
  // {
  //   std::cout<<h_bucket_ind[i]<<" ";
  // }
  // std::cout<<std::endl;
  // std::cout<<"points inds"<<std::endl;
  // for (int i = 0; i < size * (nof_bms+1); i++)
  // {
  //   std::cout<<h_point_ind[i]<<" ";
  // }
  // std::cout<<std::endl;

  // std::cout<<"pure buckets inds"<<std::endl;
  // for (int i = 0; i < size * (nof_bms+1); i++)
  // {
  //   std::cout<<h_bucket_ind[i]%(1<<(c-1))<<" ";
  // }
  // std::cout<<std::endl;
  // std::cout<<"pure points inds"<<std::endl;
  // for (int i = 0; i < size * (nof_bms+1); i++)
  // {
  //   std::cout<<h_point_ind[i]%(1<<31)<<" ";
  // }
  // std::cout<<std::endl;
                                                    

  //sort indices - the indices are sorted from smallest to largest in order to group together the points that belong to each bucket
  unsigned *sort_indices_temp_storage{};
  size_t sort_indices_temp_storage_bytes;
  hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + size, bucket_indices,
                                 point_indices + size, point_indices, size);

  hipMalloc(&sort_indices_temp_storage, sort_indices_temp_storage_bytes);
  for (unsigned i = 0; i < nof_bms; i++) {
    unsigned offset_out = i * size;
    unsigned offset_in = offset_out + size;
    hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + offset_in,
                                  bucket_indices + offset_out, point_indices + offset_in, point_indices + offset_out, size);
  }
  hipFree(sort_indices_temp_storage);

  //find bucket_sizes
  unsigned *single_bucket_indices;
  unsigned *bucket_sizes;
  unsigned *nof_buckets_to_compute;
  hipMalloc(&single_bucket_indices, sizeof(unsigned)*nof_buckets);
  hipMalloc(&bucket_sizes, sizeof(unsigned)*nof_buckets);
  hipMalloc(&nof_buckets_to_compute, sizeof(unsigned));
  unsigned *encode_temp_storage{};
  size_t encode_temp_storage_bytes = 0;
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                        nof_buckets_to_compute, nof_bms*size);
  hipMalloc(&encode_temp_storage, encode_temp_storage_bytes);
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                        nof_buckets_to_compute, nof_bms*size);
  hipFree(encode_temp_storage);

  //get offsets - where does each new bucket begin
  unsigned* bucket_offsets;
  hipMalloc(&bucket_offsets, sizeof(unsigned)*nof_buckets);
  unsigned* offsets_temp_storage{};
  size_t offsets_temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, nof_buckets);
  hipMalloc(&offsets_temp_storage, offsets_temp_storage_bytes);
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, nof_buckets);
  hipFree(offsets_temp_storage);

  //sort by bucket sizes
  unsigned* sorted_bucket_sizes;
  unsigned* sorted_bucket_offsets;
  unsigned* sorted_single_bucket_indices;
  hipMalloc(&sorted_bucket_sizes, sizeof(unsigned)*nof_buckets);
  hipMalloc(&sorted_bucket_offsets, sizeof(unsigned)*nof_buckets);
  hipMalloc(&sorted_single_bucket_indices, sizeof(unsigned)*nof_buckets);
  unsigned* sort_offsets_temp_storage{};
  size_t sort_offsets_temp_storage_bytes = 0;
  unsigned* sort_single_temp_storage{};
  size_t sort_single_temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairsDescending(sort_offsets_temp_storage, sort_offsets_temp_storage_bytes, bucket_sizes,
    sorted_bucket_sizes, bucket_offsets, sorted_bucket_offsets, nof_buckets);
  hipcub::DeviceRadixSort::SortPairsDescending(sort_single_temp_storage, sort_single_temp_storage_bytes, bucket_sizes,
    sorted_bucket_sizes, single_bucket_indices, sorted_single_bucket_indices, nof_buckets);
  hipMalloc(&sort_offsets_temp_storage, sort_offsets_temp_storage_bytes);
  hipMalloc(&sort_single_temp_storage, sort_single_temp_storage_bytes);
  hipcub::DeviceRadixSort::SortPairsDescending(sort_offsets_temp_storage, sort_offsets_temp_storage_bytes, bucket_sizes,
    sorted_bucket_sizes, bucket_offsets, sorted_bucket_offsets, nof_buckets);
  hipcub::DeviceRadixSort::SortPairsDescending(sort_single_temp_storage, sort_single_temp_storage_bytes, bucket_sizes,
    sorted_bucket_sizes, single_bucket_indices, sorted_single_bucket_indices, nof_buckets);
  hipFree(sort_offsets_temp_storage);
  hipFree(sort_single_temp_storage);
  

  //launch the accumulation kernel with maximum threads
  NUM_THREADS = 1 << 8;
  // NUM_THREADS = 1 << 5;
  NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
  // accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, point_indices, 
                                                        //  d_points, nof_buckets, nof_buckets_to_compute, c+bm_bitsize);                                              
  accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, sorted_bucket_offsets, sorted_bucket_sizes, sorted_single_bucket_indices, point_indices, 
                                                         d_points, nof_buckets, nof_buckets_to_compute, c+bm_bitsize, c);                   
  // accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, sorted_bucket_offsets, sorted_bucket_sizes, sorted_single_bucket_indices, point_indices, 
  //                                                        d_points, nof_buckets, nof_buckets_to_compute, c-1+bm_bitsize);                                              
                                                         hipDeviceSynchronize();
                                                         printf("cuda error %u",hipGetLastError());

//   hipDeviceSynchronize();
// std::vector<P> h_buckets;
//   h_buckets.reserve(nof_buckets);
//     hipMemcpy(h_buckets.data(), buckets, sizeof(P) * nof_buckets, hipMemcpyDeviceToHost);
//     std::cout<<"buckets accumulated"<<std::endl;
//     for (unsigned i = 0; i < nof_buckets; i++)
//     {
//       std::cout<<h_buckets[i]<<" ";
//     }
//     std::cout<<std::endl;
  #ifdef SSM_SUM
    //sum each bucket
    NUM_THREADS = 1 << 10;
    NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
    ssm_buckets_kernel<fake_point, fake_scalar><<<NUM_BLOCKS, NUM_THREADS>>>(buckets, single_bucket_indices, nof_buckets, c);
   
    //sum each bucket module
    P* final_results;
    hipMalloc(&final_results, sizeof(P) * nof_bms);
    NUM_THREADS = 1<<c;
    NUM_BLOCKS = nof_bms;
    sum_reduction_kernel<<<NUM_BLOCKS,NUM_THREADS>>>(buckets, final_results);
  #endif

  #ifdef BIG_TRIANGLE
    P* final_results;
    hipMalloc(&final_results, sizeof(P) * nof_bms);
    //launch the bucket module sum kernel - a thread for each bucket module
    NUM_THREADS = nof_bms;
    NUM_BLOCKS = 1;
    big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, final_results, nof_bms, c-1); //sighed digits
    // big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, final_results, nof_bms, c); 
  #endif
  hipDeviceSynchronize();
  printf("cuda error %u",hipGetLastError());

  // hipDeviceSynchronize();
  //   std::vector<P> h_final_results;
  //   h_final_results.reserve(nof_bms);
  //   hipMemcpy(h_final_results.data(), final_results, sizeof(P) * nof_bms, hipMemcpyDeviceToHost);
  //   std::cout<<"buckets summed"<<std::endl;
  //   for (unsigned i = 0; i < nof_bms; i++)
  //   {
  //     std::cout<<h_final_results[i]<<" ";
  //   }
  //   std::cout<<std::endl;


  P* d_final_result;
  if (!on_device)
    hipMalloc(&d_final_result, sizeof(P));

  //launch the double and add kernel, a single thread
  final_accumulation_kernel<P, S><<<1,1>>>(final_results, on_device ? final_result : d_final_result, 1, nof_bms, c);
  hipDeviceSynchronize();
  printf("cuda error %u",hipGetLastError());
  //copy final result to host
  hipDeviceSynchronize();
  if (!on_device)
    hipMemcpy(final_result, d_final_result, sizeof(P), hipMemcpyDeviceToHost);

  //free memory
  if (!on_device) {
    hipFree(d_points);
    hipFree(d_scalars);
    hipFree(d_final_result);
  }
  hipFree(buckets);
  hipFree(bucket_indices);
  hipFree(point_indices);
  hipFree(single_bucket_indices);
  hipFree(bucket_sizes);
  hipFree(nof_buckets_to_compute);
  hipFree(bucket_offsets);
  hipFree(sorted_bucket_sizes);
  hipFree(sorted_bucket_offsets);
  hipFree(sorted_single_bucket_indices);
  hipFree(final_results);
}

//this function computes msm using the bucket method
template <typename S, typename P, typename A>
void batched_bucket_method_msm(unsigned bitsize, unsigned c, S *scalars, A *points, unsigned batch_size, unsigned msm_size, P* final_results, bool on_device){

  unsigned total_size = batch_size * msm_size;
  S *d_scalars;
  A *d_points;
  if (!on_device) {
    //copy scalars and point to gpu
    hipMalloc(&d_scalars, sizeof(S) * total_size);
    hipMalloc(&d_points, sizeof(A) * total_size);
    hipMemcpy(d_scalars, scalars, sizeof(S) * total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_points, points, sizeof(A) * total_size, hipMemcpyHostToDevice);
  }
  else {
    d_scalars = scalars;
    d_points = points;
  }

  P *buckets;
  //compute number of bucket modules and number of buckets in each module
  unsigned nof_bms = bitsize/c;
  if (bitsize%c){
    nof_bms++;
  }
  unsigned msm_log_size = ceil(log2(msm_size));
  unsigned bm_bitsize = ceil(log2(nof_bms));
  unsigned nof_buckets = (nof_bms<<c);
  unsigned total_nof_buckets = nof_buckets*batch_size;
  hipMalloc(&buckets, sizeof(P) * total_nof_buckets); 

  //lanch the bucket initialization kernel with maximum threads
  unsigned NUM_THREADS = 1 << 10;
  unsigned NUM_BLOCKS = (total_nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
  initialize_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, total_nof_buckets); 

  unsigned *bucket_indices;
  unsigned *point_indices;
  hipMalloc(&bucket_indices, sizeof(unsigned) * (total_size * nof_bms + msm_size));
  hipMalloc(&point_indices, sizeof(unsigned) * (total_size * nof_bms + msm_size));

  //split scalars into digits
  NUM_THREADS = 1 << 8;
  NUM_BLOCKS = (total_size * nof_bms + msm_size + NUM_THREADS - 1) / NUM_THREADS;
  split_scalars_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(bucket_indices + msm_size, point_indices + msm_size, d_scalars, total_size, 
                                                    msm_log_size, nof_bms, bm_bitsize, c); //+size - leaving the first bm free for the out of place sort later

  //sort indices - the indices are sorted from smallest to largest in order to group together the points that belong to each bucket
  unsigned *sorted_bucket_indices;
  unsigned *sorted_point_indices;
  hipMalloc(&sorted_bucket_indices, sizeof(unsigned) * (total_size * nof_bms));
  hipMalloc(&sorted_point_indices, sizeof(unsigned) * (total_size * nof_bms));

  unsigned *sort_indices_temp_storage{};
  size_t sort_indices_temp_storage_bytes;
  hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + msm_size, sorted_bucket_indices,
                                 point_indices + msm_size, sorted_point_indices, total_size * nof_bms);
  hipMalloc(&sort_indices_temp_storage, sort_indices_temp_storage_bytes);
  hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + msm_size, sorted_bucket_indices,
                                 point_indices + msm_size, sorted_point_indices, total_size * nof_bms);
  hipFree(sort_indices_temp_storage);

  //find bucket_sizes
  unsigned *single_bucket_indices;
  unsigned *bucket_sizes;
  unsigned *total_nof_buckets_to_compute;
  hipMalloc(&single_bucket_indices, sizeof(unsigned)*total_nof_buckets);
  hipMalloc(&bucket_sizes, sizeof(unsigned)*total_nof_buckets);
  hipMalloc(&total_nof_buckets_to_compute, sizeof(unsigned));
  unsigned *encode_temp_storage{};
  size_t encode_temp_storage_bytes = 0;
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, sorted_bucket_indices, single_bucket_indices, bucket_sizes,
                                        total_nof_buckets_to_compute, nof_bms*total_size);
  hipMalloc(&encode_temp_storage, encode_temp_storage_bytes);
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, sorted_bucket_indices, single_bucket_indices, bucket_sizes,
                                        total_nof_buckets_to_compute, nof_bms*total_size);
  hipFree(encode_temp_storage);

  //get offsets - where does each new bucket begin
  unsigned* bucket_offsets;
  hipMalloc(&bucket_offsets, sizeof(unsigned)*total_nof_buckets);
  unsigned* offsets_temp_storage{};
  size_t offsets_temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, total_nof_buckets);
  hipMalloc(&offsets_temp_storage, offsets_temp_storage_bytes);
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, total_nof_buckets);
  hipFree(offsets_temp_storage);

  //launch the accumulation kernel with maximum threads
  NUM_THREADS = 1 << 8;
  NUM_BLOCKS = (total_nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
  accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, sorted_point_indices,
                                                        d_points, nof_buckets, total_nof_buckets_to_compute, c+bm_bitsize);

  #ifdef SSM_SUM
    //sum each bucket
    NUM_THREADS = 1 << 10;
    NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
    ssm_buckets_kernel<P, S><<<NUM_BLOCKS, NUM_THREADS>>>(buckets, single_bucket_indices, nof_buckets, c);
   
    //sum each bucket module
    P* final_results;
    hipMalloc(&final_results, sizeof(P) * nof_bms);
    NUM_THREADS = 1<<c;
    NUM_BLOCKS = nof_bms;
    sum_reduction_kernel<<<NUM_BLOCKS,NUM_THREADS>>>(buckets, final_results);
  #endif

  #ifdef BIG_TRIANGLE
    P* bm_sums;
    hipMalloc(&bm_sums, sizeof(P) * nof_bms * batch_size);
    //launch the bucket module sum kernel - a thread for each bucket module
    NUM_THREADS = 1<<8;
    NUM_BLOCKS = (nof_bms*batch_size + NUM_THREADS - 1) / NUM_THREADS;
    big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, bm_sums, nof_bms*batch_size, c);
  #endif

  P* d_final_results;
  if (!on_device)
    hipMalloc(&d_final_results, sizeof(P)*batch_size);

  //launch the double and add kernel, a single thread for each msm
  NUM_THREADS = 1<<8;
  NUM_BLOCKS = (batch_size + NUM_THREADS - 1) / NUM_THREADS;
  final_accumulation_kernel<P, S><<<NUM_BLOCKS,NUM_THREADS>>>(bm_sums, on_device ? final_results : d_final_results, batch_size, nof_bms, c);

  //copy final result to host
  hipDeviceSynchronize();
  if (!on_device)
    hipMemcpy(final_results, d_final_results, sizeof(P)*batch_size, hipMemcpyDeviceToHost);

  //free memory
  if (!on_device) {
    hipFree(d_points);
    hipFree(d_scalars);
    hipFree(d_final_results);
  }
  hipFree(buckets);
  hipFree(bucket_indices);
  hipFree(point_indices);
  hipFree(sorted_bucket_indices);
  hipFree(sorted_point_indices);
  hipFree(single_bucket_indices);
  hipFree(bucket_sizes);
  hipFree(total_nof_buckets_to_compute);
  hipFree(bucket_offsets);
  hipFree(bm_sums);

}


//this kernel converts affine points to projective points
//each thread deals with a single point
template <typename P, typename A>
__global__ void to_proj_kernel(A* affine_points, P* proj_points, unsigned N){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) proj_points[tid] = P::from_affine(affine_points[tid]);
}

//the function computes msm using ssm
template <typename S, typename P, typename A>
void short_msm(S *h_scalars, A *h_points, unsigned size, P* h_final_result){ //works up to 2^8
  S *scalars;
  A *a_points;
  P *p_points;
  P *results;

  hipMalloc(&scalars, sizeof(S) * size);
  hipMalloc(&a_points, sizeof(A) * size);
  hipMalloc(&p_points, sizeof(P) * size);
  hipMalloc(&results, sizeof(P) * size);

  //copy inputs to device
  hipMemcpy(scalars, h_scalars, sizeof(S) * size, hipMemcpyHostToDevice);
  hipMemcpy(a_points, h_points, sizeof(A) * size, hipMemcpyHostToDevice);

  //convert to projective representation and multiply each point by its scalar using single scalar multiplication
  unsigned NUM_THREADS = size;
  to_proj_kernel<<<1,NUM_THREADS>>>(a_points, p_points, size);
  ssm_kernel<<<1,NUM_THREADS>>>(scalars, p_points, results, size);

  P *final_result;
  hipMalloc(&final_result, sizeof(P));

  //assuming msm size is a power of 2
  //sum all the ssm results
  NUM_THREADS = size;
  sum_reduction_kernel<<<1,NUM_THREADS>>>(results, final_result);

  //copy result to host
  hipDeviceSynchronize();
  hipMemcpy(h_final_result, final_result, sizeof(P), hipMemcpyDeviceToHost);

  //free memory
  hipFree(scalars);
  hipFree(a_points);
  hipFree(p_points);
  hipFree(results);
  hipFree(final_result);

}

//the function computes msm on the host using the naive method
template <typename A, typename S, typename P>
void reference_msm(S* scalars, A* a_points, unsigned size){
  
  P *points = new P[size];
  // P points[size];
  for (unsigned i = 0; i < size ; i++)
  {
    points[i] = P::from_affine(a_points[i]);
  }

  P res = P::zero();
  
  for (unsigned i = 0; i < size; i++)
  {
    res = res + scalars[i]*points[i];
  }

  std::cout<<"reference results"<<std::endl;
  std::cout<<P::to_affine(res)<<std::endl;
  
}

unsigned get_optimal_c(const unsigned size) {
  if (size < 17)
    return 1;
  // return 17;
  return ceil(log2(size))-4;
}

//this function is used to compute msms of size larger than 256
template <typename S, typename P, typename A>
void large_msm(S* scalars, A* points, unsigned size, P* result, bool on_device){
  unsigned c = get_optimal_c(size);
  // unsigned c = 4;
  // unsigned bitsize = 32;
  unsigned bitsize = 253; //get from field
  bucket_method_msm(bitsize, c, scalars, points, size, result, on_device);
}

// this function is used to compute a batches of msms of size larger than 256
template <typename S, typename P, typename A>
void batched_large_msm(S* scalars, A* points, unsigned batch_size, unsigned msm_size, P* result, bool on_device){
  unsigned c = get_optimal_c(msm_size);
  // unsigned c = 6;
  // unsigned bitsize = 32;
  unsigned bitsize = 255;
  batched_bucket_method_msm(bitsize, c, scalars, points, batch_size, msm_size, result, on_device);
}
#endif
