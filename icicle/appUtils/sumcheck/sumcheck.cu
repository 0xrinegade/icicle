#include "hip/hip_runtime.h"


#define SHMEM_SIZE 64
#define MAX_SHMEM_LOG_SIZE 6

#include "../ntt/kernel_ntt.cu"
// static inline __device__ uint32_t bit_rev(uint32_t num, uint32_t log_size) { return __brev(num) >> (32 - log_size); }

// template <typename S>
// __global__ void inplace_rbo(S* arr, int size){
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	S temp = arr[tid];
// 	arr[tid] = arr[bit_rev(tid)];
// 	arr[bit_rev(tid)] = temp;
// }

template <typename S>
__global__ void mult_and_reduce(S *v, S *v_r, S alpha, int nof_results, int jump_size) {
	// Allocate shared memory
	__shared__ S partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements AND do first add of reduction
	// Vector now 2x as long as number of threads, so scale i
	int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	// Store first partial result instead of just the elements
	// partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];
	// partial_sum[threadIdx.x] = (S::one() - alpha) * v[2*i] + alpha * v[2*i+1] + (S::one() - alpha) * v[2*(i + blockDim.x)] + alpha * v[2*(i + blockDim.x)+1];
	S e1 = v[i] + alpha * (v[i+jump_size] - v[i]);
	S e2 = v[(i + blockDim.x)] + alpha * (v[(i + blockDim.x)+jump_size] - v[i + blockDim.x]);
	// S e1 = v[2*i] + (v[2*i+1] - v[2*i]);
	// S e2 = v[2*(i + blockDim.x)] + (v[2*(i + blockDim.x)+1] - v[2*(i + blockDim.x)]);
	// partial_sum[threadIdx.x] = v[2*i] + v[2*(i + blockDim.x)] + alpha * (v[2*i+1] - v[2*i] + v[2*(i + blockDim.x)+1] - v[2*(i + blockDim.x)]);
	partial_sum[threadIdx.x] = e1 + e2;
	// __syncthreads();
	v[i] = e1;
	v[i + blockDim.x] = e2;
	// for (int j = 0; j < 2; j++)
	// {
	// 	partial_sum[threadIdx.x] = partial_sum[threadIdx.x] * partial_sum[threadIdx.x];
	// }
	
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
	// for (int s = blockDim.x / 2; s > 1; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] = partial_sum[threadIdx.x] + partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	// if (threadIdx.x < nof_results) {
	if (threadIdx.x == 0) {
		// printf("debug tid %d, val %d\n", threadIdx.x, partial_sum[threadIdx.x]);
		// v_r[nof_results*blockIdx.x + threadIdx.x] = partial_sum[threadIdx.x];
		v_r[blockIdx.x] = partial_sum[0];
	}
}


template <typename S>
__global__ void sum_reduction(S *v, S *v_r, int stride) {
	// Allocate shared memory
	__shared__ S partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements AND do first add of reduction
	// Vector now 2x as long as number of threads, so scale i
	int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	// Store first partial result instead of just the elements
	partial_sum[threadIdx.x] = v[i*stride] + v[(i + blockDim.x)*stride];
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
	// for (int s = blockDim.x / 2; s > 1; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] = partial_sum[threadIdx.x] + partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	// if (threadIdx.x < nof_results) {
	if (threadIdx.x == 0) {
		// printf("debug tid %d, val %d\n", threadIdx.x, partial_sum[threadIdx.x]);
		// v_r[blockIdx.x] = partial_sum[0];
		v_r[i*stride] = partial_sum[0];
		// v_r[nof_results*blockIdx.x + threadIdx.x] = partial_sum[threadIdx.x];
	}
}

template <typename S>
__global__ void update_evals_kernel(S* evals, S alpha, int poly_size, int poly_shift, int nof_ploys){
  int threads_per_poly = poly_size/2;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= threads_per_poly*nof_ploys) return;
	int poly_id = tid / threads_per_poly;
	int eval_id = tid % threads_per_poly;
  // evals[tid] = (S::one() - alpha) * evals[2*tid] + alpha * evals[2*tid+1];
  // evals[tid] =  evals[2*tid] + (evals[2*tid+1] - evals[2*tid]);
	// if (tid==0) printf("%d, %d, %d, %d, %d\n", poly_size, poly_id, eval_id, poly_id*poly_size*2+eval_id, poly_id*poly_size*2+eval_id+poly_size);
	// if (tid==0) printf("what12 %d %d\n",evals[poly_id*poly_size*2 + eval_id], evals[poly_id*poly_size*2 + eval_id+poly_size]);
  evals[poly_id*poly_shift + eval_id] =  evals[poly_id*poly_shift+eval_id] + alpha * (evals[poly_id*poly_shift+eval_id+threads_per_poly] - evals[poly_id*poly_shift+eval_id]);
	// if (tid==0) printf("what %d\n",evals[poly_id*poly_size*2 + eval_id]);
  // evals[tid] = (1 - alpha) * evals[2*tid] + alpha * evals[2*tid+1];
}

template <typename S>
void accumulate(S* in, S* out, int log_size, int nof_results, hipStream_t stream){
  int nof_steps = (log_size - 1) / MAX_SHMEM_LOG_SIZE;
  int last_step_size = (log_size - 1) % MAX_SHMEM_LOG_SIZE;
	// printf("a nof steps %d last size %d\n", nof_steps, last_step_size);
  for (int i = 0; i < nof_steps; i++)
  {
    sum_reduction<<<(1<<(log_size - 1 - (MAX_SHMEM_LOG_SIZE)*(i+1))) * nof_results, SHMEM_SIZE/2,0,stream>>>(i? out : in, out, 1<<(MAX_SHMEM_LOG_SIZE*i));
		// printf("a nof blocks %d\n", 1<<(log_size -(MAX_SHMEM_LOG_SIZE)*(i+1)));
		// hipDeviceSynchronize();
  	// printf("cuda err %d\n", hipGetLastError());
  }
  if (last_step_size) sum_reduction<<<nof_results, 1<<(last_step_size-1), 0,stream>>>(nof_steps? out : in, out, 1<<(MAX_SHMEM_LOG_SIZE*nof_steps));
	// hipDeviceSynchronize();
  // printf("cuda err last %d\n", hipGetLastError());
}

// template <typename S>
// void accumulate(S* in, S* out, int log_size, int nof_results, hipStream_t stream){
//   int nof_steps = (log_size - 1) / MAX_SHMEM_LOG_SIZE;
//   int last_step_size = (log_size - 1) % MAX_SHMEM_LOG_SIZE;
// 	// printf("a nof steps %d last size %d\n", nof_steps, last_step_size);
//   for (int i = 0; i < nof_steps; i++)
//   {
//     sum_reduction<<<(1<<(log_size - 1 - (MAX_SHMEM_LOG_SIZE)*(i+1))) * nof_results, SHMEM_SIZE/2,0,stream>>>(i? out : in, out, 1);
// 		// printf("a nof blocks %d\n", 1<<(log_size -(MAX_SHMEM_LOG_SIZE)*(i+1)));
// 		// hipDeviceSynchronize();
//   	// printf("cuda err %d\n", hipGetLastError());
//   }
//   if (last_step_size) sum_reduction<<<nof_results, 1<<(last_step_size-1), 0,stream>>>(nof_steps? out : in, out, 1);
// 	// hipDeviceSynchronize();
//   // printf("cuda err last %d\n", hipGetLastError());
// }

template <typename S>
void mult_and_accumulate(S* in, S* out, int log_size, S alpha, int nof_results, hipStream_t stream){
  int nof_steps = (log_size - 1) / MAX_SHMEM_LOG_SIZE;
  int last_step_size = (log_size - 1) % MAX_SHMEM_LOG_SIZE;
	// printf("m nof steps %d last size %d\n", nof_steps, last_step_size);
  for (int i = 0; i < nof_steps; i++)
  {
		if (i) sum_reduction<<<1<<(log_size -(MAX_SHMEM_LOG_SIZE)*(i+1)), SHMEM_SIZE/2,0,stream>>>(i? out : in, out, nof_results);
    else mult_and_reduce<<<1<<(log_size -(MAX_SHMEM_LOG_SIZE)*(i+1)), SHMEM_SIZE/2,0,stream>>>(i? out : in, out, alpha, nof_results, 1<<log_size);
		// if (i) printf("r nof blocks %d\n", 1<<(log_size-(MAX_SHMEM_LOG_SIZE)*(i+1)));
		// else printf("m nof blocks %d\n", 1<<(log_size-(MAX_SHMEM_LOG_SIZE)*(i+1)));
		// hipDeviceSynchronize();
  	// printf("cuda err %d\n", hipGetLastError());
  }
  if (last_step_size) {
		if (nof_steps) sum_reduction<<<2, 1<<(last_step_size-1), 0,stream>>>(nof_steps? out : in, out, nof_results);
		else mult_and_reduce<<<2, 1<<(last_step_size-1), 0,stream>>>(nof_steps? out : in, out, alpha, nof_results, 1<<(last_step_size+1));
		// if (nof_steps) printf("r last");
		// else printf("m last");
	} 
	// hipDeviceSynchronize();
  // printf("cuda err last %d\n", hipGetLastError());
}

template <typename S>
 __launch_bounds__(1)
__global__ void add_to_trace(S* trace, S* vals, int stride, int round_num, int nof_results){
	for (int i = 0; i < nof_results; i++)
	{
		trace[nof_results*round_num+1+i] = vals[i*stride];
	}
	// for (int i = 0; i < nof_results; i++)
	// {
	// 	trace[nof_results*round_num+1+i] = vals[i];
	// }
	  // trace[2*round_num+1] = vals[0];
    // trace[2*round_num+2] = vals[1];
		// printf("%d  %d\n", vals[0], vals[1]);
}

template <typename S>
// __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
__global__ void combinations_kernel3(S* in, S* out, int poly_size, int poly_shift){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= poly_size/2) return;
	S rp[4] = {S::one(), S::one(), S::one(), S::one()};
	S e1, e2;
	#pragma unroll
	for (int l = 0; l < 3; l++)
	{
	  e1 = in[l*poly_shift + tid];
	  e2 = in[l*poly_shift + tid + poly_size/2];
		rp[0] = l? rp[0]*e1 : e1; //k=0
		rp[1] = l? rp[1]*e2 : e2; //k=1
		rp[2] = l? rp[2]*(e2 + e2 - e1) : (e2 + e2 - e1); //k=2
		// rp[3] = l? rp[3]*(e1 + e1 - e2) : (e1 + e1 - e2); //k=-1
		rp[3] = l? rp[3]*(e2 + e2 + e2 - e1 - e1) : (e2 + e2 + e2 - e1 - e1); //k=3
	}
	out[tid] = rp[0];
	out[tid + 1*poly_size/2] = rp[1];
	out[tid + 2*poly_size/2] = rp[2];
	out[tid + 3*poly_size/2] = rp[3];
}

template <typename S>
// __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
__global__ void mult_and_combine3(S* in, S* out, int poly_size, S alpha){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	S rp[4] = {S::one(), S::one(), S::one(), S::one()};
	S e1, e2;
	#pragma unroll
	for (int l = 0; l < 3; l++)
	{
		e1 = in[l*poly_size + 4*tid] + alpha * (in[l*poly_size + 4*tid + 1] - in[l*poly_size + 4*tid]);
		e2 = in[l*poly_size + 4*tid + 2] + alpha * (in[l*poly_size + 4*tid + 3] - in[l*poly_size + 4*tid + 2]);
		rp[0] = rp[0]*e1;
		rp[1] = rp[1]*e2;
		rp[2] = rp[2]*(e2 + e2 - e1);
		rp[3] = rp[3]*(e2 + e2 + e2 - e1 - e1);
	}
	out[4*tid] = rp[0];
	out[4*tid+1] = rp[1];
	out[4*tid+2] = rp[2];
	out[4*tid+3] = rp[3];
}

// template <typename S, int M>
// // __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
// __global__ void combinations_kernel(S* in, S* out){
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	S rp = s::one;
// 	#pragma unroll
// 	for (int k = 0; k < M+1; k++)
// 	{
// 		#pragma unroll
// 		for (int l = 0; l < M; i++)
// 		{
// 			rp *= in[2*tid] * (1 - k) + in[2*tid + 1] * k;
// 		}
// 	}
// }

// template <typename S>
// __device__ S simple_combine(S* f){
// 	return f[0]*f[1]*f[2]
// }

template <typename S>
S my_hash(){
	S val = S::one() + S::one();
	val = val + val;
	val = val + val; 
	return val + S::one() + S::one();
}

template <typename S>
void sumcheck_alg1(S* evals, S* t, S* T, S C, int n, bool reorder, hipStream_t stream){
	if (reorder) reorder_digits_inplace_and_normalize_kernel<<<1<<(max(n-6,0)),64,0,stream>>>(evals, n, false, ntt::eRevType::NaturalToRev, false, S::one());
	// S alpha = 1;
	// S alpha = S::one();
	S alpha = my_hash<S>();
	// S alpha = S::rand_host();
  // S alpha = my_hash(/*T, C*/);
  // S rp_even, rp_odd;
  for (int p = 0; p < n-1; p++)
  {
    int nof_threads = 1<<(n-1-p);
		printf("reg nof threads %d\n", nof_threads);
    // move update kernel here and unify
    // reduction_kernel<<<nof_threads>>>(evals, t, n-p); //accumulation
    accumulate(evals, t, n-p, 2, stream); //accumulation
		// hipDeviceSynchronize();
		// printf("cuda a err %d\n", hipGetLastError());
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-1-p), p, 2);
		// hipDeviceSynchronize();
		// printf("cuda t err %d\n", hipGetLastError());
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		int NOF_THREADS = min(256,nof_threads);
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
    update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, 1<<(n-p), 0, 1); //phase 3
		// hipDeviceSynchronize();
		// printf("cuda err u %d\n", hipGetLastError());
		#ifdef DEBUG
		break;
		#endif
  }
	add_to_trace<<<1,1,0,stream>>>(T, evals, 1, n-1, 2);
}

template <typename S>
void sumcheck_alg1_unified(S* evals, S* t, S* T, S C, int n, bool reorder, hipStream_t stream){
	if (reorder) reorder_digits_inplace_and_normalize_kernel<<<1<<(max(n-6,0)),64,0,stream>>>(evals, n, false, ntt::eRevType::NaturalToRev, false, S::one());
	// S alpha = 1;
	// S alpha = S::one() + S::one();
	S alpha = my_hash<S>();
	// S alpha = S::rand_host();
  // S alpha = my_hash(/*T, C*/);
  // S rp_even, rp_odd;
  for (int p = 0; p < n-1; p++)
  // for (int p = 0; p < 2; p++)
  {
    int nof_threads = 1<<(n-1-p);
		// printf("nof threads %d\n", nof_threads);
    // move update kernel here and unify
    // reduction_kernel<<<nof_threads>>>(evals, t, n-p); //accumulation
    if (p) mult_and_accumulate(evals, t, n-p, alpha, 2, stream); //accumulation
		else accumulate(evals, t, n-p, 2, stream);
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-1-p), p, 2);
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		// int NOF_THREADS = 256;
		// int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
    // update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha); //phase 3
		#ifdef DEBUG
		if (p) break;
		#endif
  }
	#ifndef DEBUG
	update_evals_kernel<<<1, 2,0, stream>>>(evals, alpha, 4, 0, 1);
	#endif
	add_to_trace<<<1,1,0,stream>>>(T, evals, 1, n-1, 2);
}

template <typename S>
void sumcheck_alg3_poly3(S* evals, S* t, S* T, S C, int n, bool reorder, hipStream_t stream){
	if (reorder) reorder_digits_inplace_and_normalize_kernel<<<1<<(max(n-6,0)),64,0,stream>>>(evals, n, false, ntt::eRevType::NaturalToRev, false, S::one());
	// S alpha = 1;
	// S alpha = S::one();
	S alpha = my_hash<S>();
	// S alpha = S::rand_host();
  // S alpha = my_hash(/*T, C*/);
  // S rp_even, rp_odd;
  for (int p = 0; p < n; p++)
  {
    int nof_threads = 1<<(n-1-p);
		int NOF_THREADS = 64;
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
		// printf("nof threads %d\n", nof_threads);
    // move update kernel here and unify
    // reduction_kernel<<<nof_threads>>>(evals, t, n-p); //accumulation
		combinations_kernel3<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-p), 1<<n);
		hipDeviceSynchronize();
		printf("cuda err u %d\n", hipGetLastError());
		if (p != n-1) accumulate(t, t, n-p, 4, stream);
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-1-p), p, 4);
		hipDeviceSynchronize();
		printf("cuda err u %d\n", hipGetLastError());
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		nof_threads = 3<<(n-1-p);
		NOF_THREADS = 64;
		NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
    if (p != n-1) update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, 1<<(n-p), 1<<n, 3); //phase 3
		hipDeviceSynchronize();
		printf("cuda err u %d\n", hipGetLastError());
		S h_evals_temp[64*3];
		hipMemcpy(h_evals_temp, evals, sizeof(S) * (64*3), hipMemcpyDeviceToHost);
		// if (p){
		// printf("round %d evals:\n",p);
		// for (int i = 0; i < 64*3; i++)
		// {
		// 	if (i % 64 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
  }
	// update_evals_kernel<<<1, 2,0, stream>>>(evals, alpha);
	// add_to_trace<<<1,1,0,stream>>>(T, evals, 1<<n, n-1, 4);
}

template <typename S>
void sumcheck_alg3_poly3_unified(S* evals, S* t, S* T, S C, int n, hipStream_t stream){
	// S alpha = 1;
	// S alpha = S::one();
	// S alpha = S::rand_host();
  S alpha = my_hash<S>();
  // S rp_even, rp_odd;
  for (int p = 0; p < n-1; p++)
  {
    int nof_threads = 1<<(n-1-p);
		int NOF_THREADS = 64;
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
		// printf("nof threads %d\n", nof_threads);
    // move update kernel here and unify
    // reduction_kernel<<<nof_threads>>>(evals, t, n-p); //accumulation
		if (p) mult_and_combine3<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<n, alpha);
		else combinations_kernel3<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-p), 1<<n);
		accumulate(t, t, n-p, 4, stream);
		add_to_trace<<<1,1,0,stream>>>(T, t, p, 4);
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
    // update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, nof_threads); //phase 3
  }
	update_evals_kernel<<<1, 2,0, stream>>>(evals, alpha, 2);
	add_to_trace<<<1,1,0,stream>>>(T, evals, n-1, 4);
}


template <typename S>
void sumcheck_alg1_ref(S* evals, S* t, S* T, S C, int n){
  // S alpha = my_hash(/*T, C*/);
	// S alpha = 1;
	// S alpha = S::one() + S::one();
	S alpha = my_hash<S>();
  S rp_bottom, rp_top;
  for (int p = 0; p < n; p++)
  {
		// rp_even = 0; rp_odd = 0;
		rp_bottom = S::zero(); rp_top = S::zero();
		// printf("evals\n");
		// for (int i = 0; i < 1<<(n-p); i++)
		// {
		// 	printf("%d, ",evals[i]);
		// }
		// printf("\n");
		for (int i = 0; i < 1<<(n-1-p); i++)
		{
			rp_bottom = rp_bottom + evals[i];
			rp_top = rp_top + evals[i+(1<<(n-1-p))];
		}
    T[2*p+1] = rp_bottom;
    T[2*p+2] = rp_top;
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		// alpha = 1;
		// alpha = S::one();
		for (int i = 0; i < 1<<(n-1-p); i++)
		{
			t[i] = (S::one() - alpha) * evals[i] + alpha * evals[i+(1<<(n-1-p))];
			// t[i] = (1-alpha)*evals[2*i] + alpha*evals[2*i+1];
		}
		for (int i = 0; i < 1<<(n-1-p); i++)
		{
			evals[i] = t[i];
		}
  }
}

template <typename S>
void sumcheck_alg3_ref(S* evals, S* t, S* T, S C, int n){
  // S alpha = my_hash(/*T, C*/);
	// S alpha = 1;
	// S alpha = S::one() + S::one();
	S alpha = my_hash<S>();
  
  for (int p = 0; p < n; p++)
  {

		// rp_even = 0; rp_odd = 0;
		// printf("evals\n");
		// for (int i = 0; i < 1<<(n-p); i++)
		// {
		// 	printf("%d, ",evals[i]);
		// }
		// printf("\n");
		for (int i = 0; i < 1<<(n-1-p); i++)
		{
			S rp[4] = {S::one(), S::one(), S::one(), S::one()};
			for (int l = 0; l < 3; l++)
			{
				S e1 = evals[(l<<(n-p)) + i];
				S e2 = evals[(l<<(n-p)) + i + (1<<(n-1-p))];
				rp[0] = l? rp[0]*e1 : e1; //k=0
				rp[1] = l? rp[1]*e2 : e2; //k=1
				rp[2] = l? rp[2]*(e2 + e2 - e1) : (e2 + e2 - e1); //k=2
				// rp[3] = l? rp[3]*(e1 + e1 - e2) : (e1 + e1 - e2); //k=-1
				rp[3] = l? rp[3]*(e2 + e2 + e2 - e1 - e1) : (e2 + e2 + e2 - e1 - e1); //k=3
			}
			T[4*p+1] = T[4*p+1] + rp[0];
			T[4*p+2] = T[4*p+2] + rp[1];
			T[4*p+3] = T[4*p+3] + rp[2];
			T[4*p+4] = T[4*p+4] + rp[3];
		}
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		// alpha = 1;
		// alpha = S::one();
		for (int l = 0; l < 3; l++)
		{
			for (int i = 0; i < 1<<(n-1-p); i++)
			{
				t[(l<<(n-1-p)) + i] = (S::one() - alpha) * evals[(l<<(n-p)) + i] + alpha * evals[(l<<(n-p)) + i + (1<<(n-1-p))];
				// t[i] = (1-alpha)*evals[2*i] + alpha*evals[2*i+1];
			}
		}
// 		if (!p)
// {		printf("ref round %d evals:\n",p);
// 		for (int i = 0; i < 3<<(n-1-p); i++)
// 		{
// 			std::cout << i << " " << evals[i] << std::endl;
// 		}}
		for (int i = 0; i < 3<<(n-1-p); i++)
		{
			evals[i] = t[i];
		}
  }
}