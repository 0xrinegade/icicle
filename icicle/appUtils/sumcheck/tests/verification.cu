#include "hip/hip_runtime.h"
#define CURVE_ID BLS12_381

#include "primitives/field.cuh"
#include "primitives/projective.cuh"
#include <chrono>
#include <iostream>
#include <vector>

// #define DEBUG
#define WARMUP
#define ONLY_BENCH

#include "curves/curve_config.cuh"
#include "sumcheck/sumcheck.cu"
#include <memory>

#include "test_vecs_381.cuh"

typedef curve_config::scalar_t test_scalar;

void random_samples(test_scalar* res, uint32_t count)
{
  for (int i = 0; i < count; i++)
    res[i] = i < 1000 ? test_scalar::rand_host() : res[i - 1000];
}

void incremental_values(test_scalar* res, uint32_t count)
{
  for (int i = 0; i < count; i++) {
    res[i] = i ? res[i - 1] + test_scalar::one() : test_scalar::one();
    // res[i] = i ? i%8==0? res[i - 1] + test_scalar::one() : res[i-1] : test_scalar::one();
    // res[i] = i%2? test_scalar::one() : test_scalar::one()+test_scalar::one();
    // res[i] = i%2? res[i - 1] : i? res[i - 1] + test_scalar::one() + test_scalar::one() : test_scalar::one() + test_scalar::one();
    // res[i] = test_scalar::one();
  }
}

int main(){

  //decleration
  test_scalar *d_transcript;
  test_scalar *d_evals;
  test_scalar *d_temp;
  test_scalar *d_transcript2;
  test_scalar *d_evals2;
  test_scalar *d_temp2;
  test_scalar C;

  hipEvent_t gpu_start, gpu_stop;
  float gpu_time;

  
  bool verify_cpu = false;
  bool use_test_vecs = verify_cpu? true : false;

  int n = 24;
  int polys = 1;
  int size = polys << n;
  int trans_size = (polys+1)*n +1;
  bool reorder = false;

  printf("Running %d polys of log2 size %d\n", polys, n);

  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  //allocation
  auto h_evals = std::make_unique<test_scalar[]>(size);
  auto h_evals_debug_ref = std::make_unique<test_scalar[]>(size);
  auto h_evals_debug_unif = std::make_unique<test_scalar[]>(size);
  auto h_temp = std::make_unique<test_scalar[]>(size);
  auto h_transcript = std::make_unique<test_scalar[]>(trans_size);
  auto h_transcript_ref = std::make_unique<test_scalar[]>(trans_size);
  
  hipMalloc(&d_transcript, sizeof(test_scalar) * (trans_size));
  hipMalloc(&d_evals, sizeof(test_scalar) * size);
  hipMalloc(&d_temp, sizeof(test_scalar) * size);
  hipMalloc(&d_transcript2, sizeof(test_scalar) * (trans_size));
  hipMalloc(&d_evals2, sizeof(test_scalar) * size);
  hipMalloc(&d_temp2, sizeof(test_scalar) * size);
  
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_stop);

  //input init

  if (polys == 1){
    if (use_test_vecs){
      if (n==3){
        // reorder=true;
        for (int i = 0; i < size; i++) {
          h_evals[i] = test_scalar{input3.storages[i]};  
        }
        for (int i = 0; i < trans_size; i++) {
          h_transcript_ref[i] = test_scalar{trans3.storages[i]};  
        }
        C = test_scalar{c3};
        h_transcript[0] = h_transcript_ref[0];
      }
      else if (n==18){
        // reorder=true;
        for (int i = 0; i < size; i++) {
          h_evals[i] = test_scalar{input18.storages[i]};  
        }
        for (int i = 0; i < trans_size; i++) {
          h_transcript_ref[i] = test_scalar{trans18.storages[i]};  
        }
        C = test_scalar{c18};
        h_transcript[0] = h_transcript_ref[0];
      }
      else{
        printf("size not supported in test vecs\n");
        return 1;
      }
    }
    else{
      // random_samples(h_evals.get(), size);
      incremental_values(h_evals.get(), size);
      C = test_scalar::rand_host();
      h_transcript[0] = test_scalar::rand_host();
      h_transcript_ref[0] = h_transcript[0];
    }
  }

  
  if (polys == 3){
    if (use_test_vecs){
      if (n==3){
        // reorder=true;
        for (int i = 0; i < size; i++) {
          h_evals[i] = test_scalar{input3poly3.storages[i]};  
        }
        for (int i = 0; i < trans_size; i++) {
          h_transcript_ref[i] = test_scalar{trans3poly3.storages[i]};  
        }
        C = test_scalar{c3poly3};
        h_transcript[0] = h_transcript_ref[0];
      }
      else if (n==10){
        // reorder=true;
        for (int i = 0; i < size; i++) {
          h_evals[i] = test_scalar{input10poly3.storages[i]};  
        }
        for (int i = 0; i < trans_size; i++) {
          h_transcript_ref[i] = test_scalar{trans10poly3.storages[i]};  
        }
        C = test_scalar{c10poly3};
        h_transcript[0] = h_transcript_ref[0];
      }
      else{
        printf("size not supported in test vecs\n");
        return 1;
      }
    }
    else {
      // random_samples(h_evals.get(), size);
      incremental_values(h_evals.get(), size);
      C = test_scalar::rand_host();
      h_transcript[0] = test_scalar::rand_host();
      h_transcript_ref[0] = h_transcript[0];
    }
  }

  hipMemcpy(d_evals, h_evals.get(), sizeof(test_scalar) * size, hipMemcpyHostToDevice);
  hipMemcpy(d_transcript, h_transcript.get(), sizeof(test_scalar), hipMemcpyHostToDevice);

#ifdef WARMUP
  //warm up run
  // sumcheck_alg1(d_evals, d_temp, d_transcript, C, n, reorder, stream1);
  // hipMemcpy(h_evals_debug_ref.get(), d_evals, sizeof(test_scalar) * (size), hipMemcpyDeviceToHost);
  // sumcheck_alg1_unified(d_evals, d_temp, d_transcript, C, n, reorder, stream1);
  // sumcheck_alg3_poly3(d_evals, d_temp, d_transcript, C, n, reorder, stream1);
  // sumcheck_alg3_poly3_unified(d_evals, d_temp, d_transcript, C, n, stream1);
  // sumcheck_alg1(d_evals2, d_temp2, d_transcript2, C, n, stream2);
  sumcheck_generic_unified(d_evals, d_temp, d_transcript, C, n, polys, stream1);
  hipDeviceSynchronize();
  hipMemcpy(d_evals, h_evals.get(), sizeof(test_scalar) * size, hipMemcpyHostToDevice);
#endif

  //run
  hipEventRecord(gpu_start, 0);
  // if (verify_cpu && polys == 1) sumcheck_alg1_ref(h_evals.get(), h_temp.get(), h_transcript.get(), C, n);
  // if (verify_cpu && polys == 3) sumcheck_alg3_ref(h_evals.get(), h_temp.get(), h_transcript.get(), C, n);
  // if (polys == 1) sumcheck_alg1(d_evals, d_temp, d_transcript, C, n, reorder, stream1);
  // if (polys == 1) sumcheck_alg1_unified(d_evals, d_temp, d_transcript, C, n, reorder, stream1);
  // hipMemcpy(h_evals_debug_unif.get(), d_evals, sizeof(test_scalar) * (size), hipMemcpyDeviceToHost);
  // if (polys == 3) sumcheck_alg3_poly3(d_evals, d_temp, d_transcript, C, n, reorder, stream1);
  // if (polys == 3) sumcheck_alg3_poly3_unified(d_evals, d_temp, d_transcript, C, n, stream1);
  sumcheck_generic_unified(d_evals, d_temp, d_transcript, C, n, polys, stream1);
  // sumcheck_alg1(d_evals2, d_temp2, d_transcript2, C, n, stream2);
  hipEventRecord(gpu_stop, 0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&gpu_time, gpu_start, gpu_stop);
  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);

  #ifndef ONLY_BENCH
  //run reference
  auto cpu_start = std::chrono::high_resolution_clock::now();
  if (!use_test_vecs && polys == 1) sumcheck_alg1_ref(h_evals.get(), h_temp.get(), h_transcript_ref.get(), C, n);
  if (!use_test_vecs && polys == 3) sumcheck_alg3_ref(h_evals.get(), h_temp.get(), h_transcript_ref.get(), C, n);
  auto cpu_stop = std::chrono::high_resolution_clock::now();
  auto cpu_time = std::chrono::duration_cast<std::chrono::microseconds>(cpu_stop - cpu_start).count();

  //verify
  if (!verify_cpu) hipMemcpy(h_transcript.get(), d_transcript, sizeof(test_scalar) * (trans_size), hipMemcpyDeviceToHost);
  
  bool success = true;
  #ifdef DEBUG
  for (int i = 0; i < size; i++) {
    if (h_evals_debug_ref[i] != h_evals_debug_unif[i]) {
      success = false;
      std::cout << i << " ref " << h_evals_debug_ref[i] << " != " << h_evals_debug_unif[i] << std::endl;
    } else {
      std::cout << i << " ref " << h_evals_debug_ref[i] << " == " << h_evals_debug_unif[i] << std::endl;
    }
  }
  printf("\n");
  #endif
  for (int i = 0; i < trans_size; i++) {
    if (h_transcript[i] != h_transcript_ref[i]) {
      success = false;
      std::cout << i << " ref " << h_transcript_ref[i] << " != " << h_transcript[i] << std::endl;
    } else {
      std::cout << i << " ref " << h_transcript_ref[i] << " == " << h_transcript[i] << std::endl;
    }
  }
  const char* success_str = success ? "SUCCESS!" : "FAIL!";
  printf("%s\n", success_str);
  
  //print times
  std::cout << "CPU Runtime=" << cpu_time / 1000 << " MS" << std::endl;
  #endif
  printf("GPU Runtime=%0.3f MS\n", gpu_time);
  // printf("CPU Runtime=%0.3f MS\n", cpu_time);

  //free
  hipFree(d_evals);
  hipFree(d_temp);
  hipFree(d_transcript);


}


// int main(){

//   int evals[8] = {0,1,2,3,4,5,6,7};
//   int t[8] = {0,0,0,0,0,0,0,0};
//   int T[8] = {45,0,0,0,0,0,0,0};
//   int C = 33;
//   int n = 3;

//   int *d_evals;
//   int *d_t;
//   int *d_T;
  
//   int log_size = 17;
//   int size = 1<<log_size;
//   auto largEvals = std::make_unique<int[]>(size);
//   for (int i = 0; i < size; i++)
//   {
//     largEvals[i] = i%2? 1 : 2;
//   }
  

//   // hipMalloc(&d_evals, sizeof(int) * size);
//   // hipMemcpy(d_evals, largEvals.get(), sizeof(int) * size, hipMemcpyHostToDevice);
//   hipMalloc(&d_evals, sizeof(int) * 8);
//   hipMemcpy(d_evals, evals, sizeof(int) * 8, hipMemcpyHostToDevice);
//   hipMalloc(&d_t, sizeof(int) * 8);
//   hipMalloc(&d_T, sizeof(int) * 8);
//   hipMemcpy(d_t, t, sizeof(int) * 8, hipMemcpyHostToDevice);
//   hipMemcpy(d_T, T, sizeof(int) * 8, hipMemcpyHostToDevice);

//   for (int i = 0; i < 8; i++)
//   {
//     std::cout << evals[i] <<std::endl;
//   }
//   // accumulate(d_evals, d_evals, log_size);
//   // hipDeviceSynchronize();
//   // printf("cuda err %d\n", hipGetLastError());
//   // hipMemcpy(largEvals.get(), d_evals, sizeof(int) * size, hipMemcpyDeviceToHost);
//   // for (int i = 0; i < 8; i++)
//   // {
//   //   std::cout << largEvals[i] <<std::endl;
//   // }


//   sumcheck_alg1(d_evals, d_t, d_T, C, n);
//   // sumcheck_alg1_ref(evals, t, T, C, n);

//   // for (int i = 0; i < 8; i++)
//   // {
//   //   std::cout << T[i] <<std::endl;
//   // }

//   hipMemcpy(T, d_T, sizeof(int) * 8, hipMemcpyDeviceToHost);

//   for (int i = 0; i < 8; i++)
//   {
//     std::cout << T[i] <<std::endl;
//   }

//   return 0;
// }