#include "hip/hip_runtime.h"

#include "icicle/polynomials/polynomials.h"
#include "icicle/polynomials/default_backend/default_poly_context.h"

#include "gpu-utils/error_handler.h"
#include "hip/hip_runtime.h"
#include "icicle/ntt.h"
#include "kernels.cuh"
#include "icicle/runtime.h"
#include "icicle/errors.h"

namespace polynomials {

  /*============================== Polynomial Default-context ==============================*/

  using icicle::DefaultPolynomialContext;

  /*============================== Polynomial Default-backend ==============================*/

  template <typename C = scalar_t, typename D = C, typename I = C>
  class DefaultPolynomialBackend : public IPolynomialBackend<C, D, I>
  {
    typedef std::shared_ptr<IPolynomialContext<C, D, I>> PolyContext;
    typedef typename IPolynomialContext<C, D, I>::State State;

    int64_t* d_degree = nullptr; // used to avoid alloc/release every time

  public:
    hipStream_t m_stream; // TODO Yuval make it icicleStreamHandle
    DefaultPolynomialBackend(const icicleStreamHandle stream) : m_stream{reinterpret_cast<hipStream_t>(stream)}
    {
      ICICLE_CHECK(icicle_malloc_async((void**)&d_degree, sizeof(int64_t), m_stream));
    }
    ~DefaultPolynomialBackend() noexcept { icicle_free_async(d_degree, m_stream); }

    void from_coefficients(PolyContext p, uint64_t nof_coefficients, const C* coefficients) override
    {
      p->from_coefficients(nof_coefficients, coefficients);
    }

    void from_rou_evaluations(PolyContext p, uint64_t nof_evaluations, const I* evaluations) override
    {
      p->from_rou_evaluations(nof_evaluations, evaluations);
    }

    void clone(PolyContext out, PolyContext in) override { out->clone(*in); }

    template <typename T = C>
    T* get_context_storage_mutable(PolyContext p)
    {
      return static_cast<T*>(IPolynomialBackend<C, D, I>::get_context_storage_mutable(p));
    }

    template <typename T = C>
    const T* get_context_storage_immutable(PolyContext p) const
    {
      return static_cast<const T*>(IPolynomialBackend<C, D, I>::get_context_storage_immutable(p));
    }

    void slice(PolyContext out, PolyContext in, uint64_t offset, uint64_t stride, uint64_t size) override
    {
      assert_device_compatability({in});
      auto [in_coeffs, in_size] = in->get_coefficients();
      // size=0 means take as much as elements as there are to take
      uint64_t out_size = (size > 0) ? size : (1 + (in_size - 1 - offset) / stride);

      out->allocate(out_size, State::Coefficients, false /*=memset zeros*/);
      auto out_coeffs = get_context_storage_mutable(out);

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (out_size + NOF_THREADS - 1) / NOF_THREADS;
      slice_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(in_coeffs, out_coeffs, offset, stride, out_size);

      CHK_LAST();
    }

    void add_sub(PolyContext& res, PolyContext a, PolyContext b, bool add1_sub0)
    {
      assert_device_compatability({a, b});

      // add/sub can be done in both coefficients or evaluations, but operands must be in the same state.
      // For evaluations, same state also means same number of evaluations (and on same domain).
      // If not same state, compute in coefficients since computing in evaluations may require to interpolate a large
      // size. Consider a+b where a is degree 128 and b degree 4. In coefficients b has 4 elements but in evaluations
      // need 128.
      const bool is_same_size = a->get_nof_elements() == b->get_nof_elements();
      const bool is_same_state = a->get_state() == b->get_state();
      const auto output_state = (is_same_size && is_same_state) ? a->get_state() : State::Coefficients;
      const auto output_size = max(a->get_nof_elements(), b->get_nof_elements());

      if (State::Coefficients == output_state) {
        a->transform_to_coefficients();
        b->transform_to_coefficients();
      }
      const auto a_mem_p = get_context_storage_immutable(a);
      const auto b_mem_p = get_context_storage_immutable(b);

      res->allocate(output_size, output_state);
      auto res_mem_p = get_context_storage_mutable(res);

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (output_size + NOF_THREADS - 1) / NOF_THREADS;
      add_sub_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(
        a_mem_p, b_mem_p, a->get_nof_elements(), b->get_nof_elements(), add1_sub0, res_mem_p);

      CHK_LAST();
    }

    void add(PolyContext& res, PolyContext a, PolyContext b) override { add_sub(res, a, b, true /*=add*/); }
    void subtract(PolyContext res, PolyContext a, PolyContext b) override { add_sub(res, a, b, false /*=sub*/); }

    void multiply(PolyContext c, PolyContext a, PolyContext b) override
    {
      assert_device_compatability({a, b});

      const bool is_a_scalar = a->get_nof_elements() == 1;
      const bool is_b_scalar = b->get_nof_elements() == 1;

      // TODO: can add kernel that takes the scalar as device memory
      if (is_a_scalar) {
        return multiply(c, b, get_coeff(a, 0));
      } else if (is_b_scalar) {
        return multiply(c, a, get_coeff(b, 0));
      }

      const bool is_multiply_with_cosets = true; // TODO  Yuval: check when faster to do so.
      if (is_multiply_with_cosets) { return multiply_with_cosets(c, a, b); }
      return multiply_with_padding(c, a, b);
    }

    void multiply(PolyContext out, PolyContext p, D scalar) override
    {
      assert_device_compatability({p});

      // element wise multiplication is similar both in coefficients and evaluations (regardless of order too)
      const auto state = p->get_state();
      const auto N = p->get_nof_elements();

      auto p_elements_p =
        state == State::Coefficients ? get_context_storage_immutable<C>(p) : get_context_storage_immutable<I>(p);

      out->allocate(N, state, false /*=memset zeros*/);
      auto out_evals_p =
        state == State::Coefficients ? get_context_storage_mutable<C>(out) : get_context_storage_mutable<I>(out);

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (N + NOF_THREADS - 1) / NOF_THREADS;
      mul_scalar_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(p_elements_p, scalar, N, out_evals_p);

      CHK_LAST();
    }

    void multiply_with_padding(PolyContext c, PolyContext a, PolyContext b)
    {
      // TODO Yuval: by using the degree I can optimize the memory size and avoid redundant computations too
      const uint64_t a_N_orig = a->get_nof_elements();
      const uint64_t b_N_orig = b->get_nof_elements();
      const uint64_t N = max(a_N_orig, b_N_orig);
      const uint64_t c_N = 2 * N;

      // (1) transform a,b to 2N evaluations
      a->transform_to_evaluations(c_N, true /*=reversed*/);
      b->transform_to_evaluations(c_N, true /*=reversed*/);
      auto [a_evals_p, a_N] = a->get_rou_evaluations();
      auto [b_evals_p, b_N] = b->get_rou_evaluations();

      // (2) allocate c (c=a*b) and compute element-wise multiplication on evaluations
      c->allocate(c_N, State::EvaluationsOnRou_Reversed, false /*=memset zeros*/);
      auto c_evals_p = get_context_storage_mutable<I>(c);

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (c_N + NOF_THREADS - 1) / NOF_THREADS;
      mul_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(a_evals_p, b_evals_p, c_N, c_evals_p);

      CHK_LAST();
    }

    void multiply_with_cosets(PolyContext c, PolyContext a, PolyContext b)
    {
      const uint64_t a_N = a->get_nof_elements();
      const uint64_t b_N = b->get_nof_elements();
      const uint64_t N = max(a_N, b_N);

      // (1) transform a,b to coefficients such that both have N coefficients
      a->transform_to_coefficients(N);
      b->transform_to_coefficients(N);
      auto [a_coeff_p, _] = a->get_coefficients();
      auto [b_coeff_p, __] = b->get_coefficients();
      // (2) allocate c (c=a*b)
      const uint64_t c_N = 2 * N;
      c->allocate(c_N, State::EvaluationsOnRou_Reversed, false /*=memset zeros*/);
      auto c_evals_low_p = get_context_storage_mutable<I>(c);
      I* c_evals_high_p = c_evals_low_p + N;

      // (3) compute NTT of a,b on coset and write to c
      auto ntt_config = default_ntt_config<C>();
      ntt_config.are_inputs_on_device = true;
      ntt_config.are_outputs_on_device = true;
      ntt_config.is_async = true;
      ntt_config.ordering = Ordering::kNR;
      ICICLE_CHECK(get_root_of_unity_from_domain<C>((uint64_t)log2(c_N), ntt_config.coset_gen));

      ICICLE_CHECK(ntt(a_coeff_p, N, NTTDir::kForward, ntt_config, c_evals_low_p));  // a_H1
      ICICLE_CHECK(ntt(b_coeff_p, N, NTTDir::kForward, ntt_config, c_evals_high_p)); // b_H1

      // (4) compute a_H1 * b_H1 inplace
      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (N + NOF_THREADS - 1) / NOF_THREADS;
      mul_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(c_evals_low_p, c_evals_high_p, N, c_evals_high_p);
      // (5) transform a,b to evaluations
      a->transform_to_evaluations(N, true /*=reversed*/);
      b->transform_to_evaluations(N, true /*=reversed*/);
      auto [a_evals_p, a_nof_evals] = a->get_rou_evaluations();
      auto [b_evals_p, b_nof_evals] = b->get_rou_evaluations();

      // (6) compute a_H0 * b_H0
      mul_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(a_evals_p, b_evals_p, N, c_evals_low_p);

      CHK_LAST();
    }

    void divide(PolyContext Q /*OUT*/, PolyContext R /*OUT*/, PolyContext a, PolyContext b) override
    {
      assert_device_compatability({a, b});

      auto [a_coeffs, a_N] = a->get_coefficients();
      auto [b_coeffs, b_N] = b->get_coefficients();

      const int64_t deg_a = degree(a);
      const int64_t deg_b = degree(b);
      ICICLE_ASSERT(deg_b >= 0) << "Polynomial division:  divide by zero polynomial";

      // init: Q=0, R=a
      Q->allocate(deg_a - deg_b + 1, State::Coefficients, true /*=memset zeros*/);
      auto Q_coeffs = get_context_storage_mutable(Q);

      //    TODO Yuval: Can do better in terms of memory allocation? deg(R) <= deg(b) by definition but it starts as
      R->allocate(a_N, State::Coefficients, false /*=memset_zeros*/);
      auto R_coeffs = get_context_storage_mutable(R);
      ICICLE_CHECK(icicle_copy_async(R_coeffs, a_coeffs, a_N * sizeof(C), m_stream));

      const C& lc_b_inv = C::inverse(get_coeff(b, deg_b)); // largest coeff of b

      int64_t deg_r = deg_a;
      while (deg_r >= deg_b) {
        // each iteration is removing the largest monomial in r until deg(r)<deg(b)
        const int NOF_THREADS = 128;
        const int NOF_BLOCKS = ((deg_r + 1) + NOF_THREADS - 1) / NOF_THREADS; // 'deg_r+1' is number of elements in R
        school_book_division_step<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(
          R_coeffs, Q_coeffs, b_coeffs, deg_r, deg_b, lc_b_inv);

        // faster than degree(R) based on the fact that degree is decreasing
        deg_r = degree_internal(R, deg_r + 1 /*size of R*/);
      }

      CHK_LAST();
    }

    void quotient(PolyContext Q, PolyContext op_a, PolyContext op_b) override
    {
      // TODO: can implement more efficiently?
      auto R = std::make_shared<DefaultPolynomialContext<C, D, I>>(m_stream);
      divide(Q, R, op_a, op_b);
    }

    void remainder(PolyContext R, PolyContext op_a, PolyContext op_b) override
    {
      // TODO: can implement more efficiently?
      auto Q = std::make_shared<DefaultPolynomialContext<C, D, I>>(m_stream);
      divide(Q, R, op_a, op_b);
    }

    void divide_by_vanishing_polynomial(PolyContext out, PolyContext numerator, uint64_t vanishing_poly_degree) override
    {
      assert_device_compatability({numerator});

      // vanishing polynomial of degree N is the polynomial V(x) such that V(r)=0 for r Nth root-of-unity.
      // For example for N=4 it vanishes on the group [1,W,W^2,W^3] where W is the 4th root of unity. In that
      // case V(x)=(x-1)(x-w)(x-w^2)(x-w^3). It can be easily shown that V(x)=x^N-1. This holds since x^N=1 on this
      // domain (since x is the Nth root of unity).

      // Note that we always represent polynomials with N elements for N a power of two. This is required for NTTs.
      // In addition we consider deg(P) to be this number of elements N even though the real degree may be lower. for
      // example 1+x-2x^2 is degree 2 but we store 4 elements and consider it degree 3.

      // when dividing a polynomial  P(x)/V(x) (The vanishing polynomial) the output is of degree deg(P)-deg(V). There
      // are three cases where V(x) divides P(x) (this is assumed since otherwise the output polynomial does not
      // exist!):
      // (1) deg(P)=2*deg(V): in that case deg(P/V)=deg(V)=N. This is an efficient case since on a domain of size N, the
      // vanishing polynomial evaluates to a constant value.
      // (2) deg(P)=deg(V)=N: in that case the output is a degree 0 polynomial.
      // polynomial (i.e. scalar). (3) general case: deg(P)>2*deg(V): in that case deg(P) is a least 4*deg(V) since N is
      // a power of two. This means that deg(P/V)=deg(P). For example deg(P)=16, deg(V)=4 --> deg(P/V)=12 ceiled to 16.

      // When computing we want to divide P(x)'s evals by V(x)'s evals. Since V(x)=0 on this domain we have to compute
      // on a coset.
      // for case (3) we must evaluate V(x) on deg(P) domain size and compute elementwise division on a coset.
      // case (1) is more efficient because we need N evaluations of V(x) on a coset. Note that V(x)=constant on a coset
      // of rou. This is because V(wu)=(wu)^N-1=W^N*u^N-1 = 1*u^N-1 (as w^N=1 for w Nth root of unity). case (2) can be
      // computed like case (1).

      const bool is_case_2N = numerator->get_nof_elements() == 2 * vanishing_poly_degree;
      const bool is_case_N = numerator->get_nof_elements() == vanishing_poly_degree;
      if (is_case_2N) {
        divide_by_vanishing_case_2N(out, numerator, vanishing_poly_degree);
      } else if (is_case_N) {
        divide_by_vanishing_case_N(out, numerator, vanishing_poly_degree);
      } else {
        divide_by_vanishing_general_case(out, numerator, vanishing_poly_degree);
      }
    }

    void divide_by_vanishing_general_case(PolyContext out, PolyContext numerator, uint64_t vanishing_poly_degree)
    {
      // General case: P(x)/V(x) where v is of degree N and p of any degree>N

      // (1) allocate vanishing polynomial in coefficients form
      // TODO Yuval: maybe instead of taking numerator memory and modiyfing it diretcly add a state for evaluations
      // on coset of rou. In that case I can remain in this state and also won't need to access input memory
      // directly
      numerator->transform_to_coefficients();
      auto numerator_coeffs = get_context_storage_mutable(numerator);
      const auto N = numerator->get_nof_elements();
      if (vanishing_poly_degree > N) {
        THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "divide_by_vanishing_polynomial(): degree is too large");
      }
      out->allocate(N, State::Coefficients, true /*=set zeros*/);
      add_monomial_inplace(out, C::zero() - C::one(), 0);         //-1
      add_monomial_inplace(out, C::one(), vanishing_poly_degree); //+x^n

      // (2) NTT on coset. Note that NTT on ROU evaluates to zeros for vanihsing polynomials by definition.
      // Therefore evaluation on coset is required to compute non-zero evaluations, which make element-wise division
      // possible
      auto out_coeffs = get_context_storage_mutable(out);
      auto ntt_config = default_ntt_config<C>();
      ntt_config.are_inputs_on_device = true;
      ntt_config.are_outputs_on_device = true;
      ntt_config.is_async = true;
      ntt_config.ordering = Ordering::kNM;
      ICICLE_CHECK(get_root_of_unity_from_domain<C>((uint64_t)log2(2 * N), ntt_config.coset_gen));

      ICICLE_CHECK(ntt(out_coeffs, N, NTTDir::kForward, ntt_config, out_coeffs));
      ICICLE_CHECK(ntt(numerator_coeffs, N, NTTDir::kForward, ntt_config, numerator_coeffs));

      // (3) element wise division
      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (N + NOF_THREADS - 1) / NOF_THREADS;
      div_element_wise_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(numerator_coeffs, out_coeffs, N, out_coeffs);

      // (4) INTT back both numerator and out
      ntt_config.ordering = Ordering::kMN;
      ICICLE_CHECK(ntt(out_coeffs, N, NTTDir::kInverse, ntt_config, out_coeffs));
      ICICLE_CHECK(ntt(numerator_coeffs, N, NTTDir::kInverse, ntt_config, numerator_coeffs));
    }

    void divide_by_vanishing_case_2N(PolyContext out, PolyContext numerator, uint64_t vanishing_poly_degree)
    {
      // in that special case the numertaor has 2N elements and output will be N elements
      if (numerator->get_nof_elements() != 2 * vanishing_poly_degree) {
        THROW_ICICLE_ERR(IcicleError_t::UndefinedError, "invalid input size. Expecting numerator to be of size 2N");
      }

      // In the case where deg(P)=2N, I can transform numerator to Reversed-evals -> The second half is
      // a reversed-coset of size N with coset-gen the 2N-th root of unity.
      const int N = vanishing_poly_degree;
      numerator->transform_to_evaluations(2 * N, true /*=reversed*/);
      // allocate output in coeffs because it will be calculated on a coset but I don't have such a state so will have
      // to INTT back to coeffs
      auto numerator_evals_reversed_p = get_context_storage_immutable<I>(numerator);
      out->allocate(N, State::Coefficients, false /*=set zeros*/);
      auto out_evals_reversed_p = get_context_storage_mutable<I>(out);

      auto ntt_config = default_ntt_config<C>();
      ICICLE_CHECK(get_root_of_unity_from_domain<C>((uint64_t)log2(2 * N), ntt_config.coset_gen));
      // compute inv(u^N-1);
      D v_coset_eval = D::inverse(D::pow(ntt_config.coset_gen, N) - D::one());

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (N + NOF_THREADS - 1) / NOF_THREADS;
      mul_scalar_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(
        numerator_evals_reversed_p + N /*second half is the reversed coset*/, v_coset_eval, N, out_evals_reversed_p);

      // INTT back from reversed evals on coset to coeffs
      ntt_config.are_inputs_on_device = true;
      ntt_config.are_outputs_on_device = true;
      ntt_config.is_async = true;
      ntt_config.ordering = Ordering::kRN;
      ntt(out_evals_reversed_p, N, NTTDir::kInverse, ntt_config, out_evals_reversed_p);

      CHK_LAST();
    }

    void divide_by_vanishing_case_N(PolyContext out, PolyContext numerator, uint64_t vanishing_poly_degree)
    {
      // in that special case the numertaor has N elements and output will be N elements
      if (numerator->get_nof_elements() != vanishing_poly_degree) {
        THROW_ICICLE_ERR(IcicleError_t::UndefinedError, "invalid input size. Expecting numerator to be of size N");
      }

      const int N = vanishing_poly_degree;
      numerator->transform_to_coefficients(N);
      auto numerator_evals_reversed_p = get_context_storage_immutable<I>(numerator);
      out->allocate(N, State::Coefficients, false /*=set zeros*/);
      auto out_evals_reversed_p = get_context_storage_mutable<I>(out);

      // (1) NTT numerator to coset evals (directly to out)
      auto ntt_config = default_ntt_config<C>();
      ICICLE_CHECK(get_root_of_unity_from_domain<C>((uint64_t)log2(2 * N), ntt_config.coset_gen));
      ntt_config.are_inputs_on_device = true;
      ntt_config.are_outputs_on_device = true;
      ntt_config.is_async = true;
      ntt_config.ordering = Ordering::kNM;
      ntt(numerator_evals_reversed_p, N, NTTDir::kForward, ntt_config, out_evals_reversed_p);

      // (2) divide by constant value (that V(x) evaluates to on the coset)
      D v_coset_eval = D::inverse(D::pow(ntt_config.coset_gen, N) - D::one());

      const int NOF_THREADS = 128;
      const int NOF_BLOCKS = (N + NOF_THREADS - 1) / NOF_THREADS;
      mul_scalar_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(
        out_evals_reversed_p, v_coset_eval, N, out_evals_reversed_p);

      // (3) INTT back from coset to coeffs
      ntt_config.are_inputs_on_device = true;
      ntt_config.are_outputs_on_device = true;
      ntt_config.is_async = true;
      ntt_config.ordering = Ordering::kMN;
      ntt(out_evals_reversed_p, N, NTTDir::kInverse, ntt_config, out_evals_reversed_p);

      CHK_LAST();
    }

    // arithmetic with monomials
    void add_monomial_inplace(PolyContext& poly, C monomial_coeff, uint64_t monomial) override
    {
      const uint64_t new_nof_elements = max(poly->get_nof_elements(), monomial + 1);
      poly->transform_to_coefficients(new_nof_elements);
      auto coeffs = get_context_storage_mutable(poly);
      add_single_element_inplace<<<1, 1, 0, m_stream>>>(coeffs + monomial, monomial_coeff);

      CHK_LAST();
    }

    void sub_monomial_inplace(PolyContext& poly, C monomial_coeff, uint64_t monomial) override
    {
      add_monomial_inplace(poly, C::zero() - monomial_coeff, monomial);
    }

    int64_t degree(PolyContext p) override { return degree_internal(p, p->get_nof_elements()); }

    // search degree starting from len, searching down (towards coeff0)
    int64_t degree_internal(PolyContext p, uint64_t len)
    {
      // TODO: parallelize kernel? Note that typically the largest coefficient is expected in the higher half since
      // memory is allocate based on #coefficients

      auto [coeff, _] = p->get_coefficients();

      int64_t h_degree;
      highest_non_zero_idx<<<1, 1, 0, m_stream>>>(coeff, len, d_degree);
      ICICLE_CHECK(icicle_copy_async(&h_degree, d_degree, sizeof(int64_t), m_stream));
      ICICLE_CHECK(icicle_stream_synchronize(m_stream)); // sync to make sure return value is copied to host

      return h_degree;
    }

  public:
    void evaluate(PolyContext p, const D* x, I* eval) override
    {
      // TODO Yuval: maybe use Horner's rule and just evaluate each domain point per thread. Alternatively Need to
      // reduce in parallel.

      auto [coeff, nof_coeff] = p->get_coefficients();

      const bool is_x_on_host = is_host_ptr(x);
      const bool is_eval_on_host = is_host_ptr(eval);

      const D* d_x = x;
      D* allocated_x = nullptr;
      if (is_x_on_host) {
        ICICLE_CHECK(icicle_malloc_async((void**)&allocated_x, sizeof(I), m_stream));
        ICICLE_CHECK(icicle_copy_async(allocated_x, x, sizeof(I), m_stream));
        d_x = allocated_x;
      }
      I* d_eval = eval;
      if (is_eval_on_host) { ICICLE_CHECK(icicle_malloc_async((void**)&d_eval, sizeof(I), m_stream)); }

      // TODO Yuval: other methods can avoid this allocation. Also for eval_on_domain() no need to reallocate every time
      I* d_tmp = nullptr;
      ICICLE_CHECK(icicle_malloc_async((void**)&d_tmp, sizeof(I) * nof_coeff, m_stream));
      const int NOF_THREADS = 32;
      const int NOF_BLOCKS = (nof_coeff + NOF_THREADS - 1) / NOF_THREADS;
      evaluate_polynomial_without_reduction<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(
        d_x, coeff, nof_coeff, d_tmp); // TODO Yuval: parallelize kernel
      dummy_reduce<<<1, 1, 0, m_stream>>>(d_tmp, nof_coeff, d_eval);

      if (is_eval_on_host) {
        ICICLE_CHECK(icicle_copy_async(eval, d_eval, sizeof(I), m_stream));
        ICICLE_CHECK(icicle_stream_synchronize(m_stream)); // sync to make sure return value is copied to host
        ICICLE_CHECK(icicle_free_async(d_eval, m_stream));
      }
      if (allocated_x) { ICICLE_CHECK(icicle_free_async(allocated_x, m_stream)); }
      ICICLE_CHECK(icicle_free_async(d_tmp, m_stream));
    }

    void evaluate_on_domain(PolyContext p, const D* domain, uint64_t size, I* evaluations /*OUT*/) override
    {
      // TODO Yuval: implement more efficiently ??
      for (uint64_t i = 0; i < size; ++i) {
        evaluate(p, &domain[i], &evaluations[i]);
      }
    }

    void evaluate_on_rou_domain(PolyContext p, uint64_t domain_log_size, I* evals /*OUT*/) override
    {
      const uint64_t poly_size = p->get_nof_elements();
      const uint64_t domain_size = 1 << domain_log_size;
      const bool is_evals_on_host = is_host_ptr(evals);

      I* d_evals = evals;
      // if evals on host, allocate memory
      if (is_evals_on_host) { ICICLE_CHECK(icicle_malloc_async((void**)&d_evals, domain_size * sizeof(I), m_stream)); }

      // If domain size is smaller the polynomial size -> transform to evals and copy the evals with stride.
      // Else, if in coeffs copy coeffs to evals mem and NTT inplace to compute the evals, else INTT to d_evals and back
      // inplace to larger domain
      const bool is_domain_size_smaller_than_poly_size = domain_size <= poly_size;
      if (is_domain_size_smaller_than_poly_size) {
        // TODO Yuval: in reversed evals, can reverse the first 'domain_size' elements to d_evals instead of
        // transforming back to evals.
        p->transform_to_evaluations();
        const auto stride = poly_size / domain_size;
        const int NOF_THREADS = 128;
        const int NOF_BLOCKS = (domain_size + NOF_THREADS - 1) / NOF_THREADS;
        slice_kernel<<<NOF_BLOCKS, NOF_THREADS, 0, m_stream>>>(
          get_context_storage_immutable<I>(p), d_evals, 0 /*offset*/, stride, domain_size);
      } else {
        ICICLE_CHECK(icicle_memset(d_evals, 0, domain_size * sizeof(I)));
        auto ntt_config = default_ntt_config<D>();
        ntt_config.are_inputs_on_device = true;
        ntt_config.are_outputs_on_device = true;
        ntt_config.is_async = true;
        // TODO Yuval: in evals I can NTT directly to d_evals without changing my state
        switch (p->get_state()) {
        case State::Coefficients: {
          // copy to evals memory and inplace NTT of domain size
          ICICLE_CHECK(icicle_copy(d_evals, get_context_storage_immutable<I>(p), poly_size * sizeof(I)));
          ntt_config.ordering = Ordering::kNN;
          ntt(d_evals, domain_size, NTTDir::kForward, ntt_config, d_evals);
        } break;
        case State::EvaluationsOnRou_Natural:
        case State::EvaluationsOnRou_Reversed: {
          const bool is_from_natrual = p->get_state() == State::EvaluationsOnRou_Natural;
          // INTT to coeffs and back to evals
          ntt_config.ordering = is_from_natrual ? Ordering::kNM : Ordering::kRN;
          ntt(get_context_storage_immutable<I>(p), poly_size, NTTDir::kInverse, ntt_config, d_evals);
          ntt_config.ordering = is_from_natrual ? Ordering::kMN : Ordering::kNN;
          ntt(d_evals, poly_size, NTTDir::kForward, ntt_config, d_evals);
        } break;
        default:
          THROW_ICICLE_ERR(IcicleError_t::UndefinedError, "Invalid state to compute evaluations");
          break;
        }
      }

      // release memory if allocated
      if (is_evals_on_host) {
        ICICLE_CHECK(icicle_copy_async(evals, d_evals, domain_size * sizeof(I), m_stream));
        ICICLE_CHECK(icicle_free_async(d_evals, m_stream));
      }

      // sync since user cannot reuse this stream so need to make sure evals are computed
      ICICLE_CHECK(icicle_stream_synchronize(m_stream)); // sync to make sure return value is copied to host

      CHK_LAST();
    }

    uint64_t copy_coeffs(PolyContext op, C* out_coeffs, uint64_t start_idx, uint64_t end_idx) override
    {
      const uint64_t nof_coeffs = op->get_nof_elements();
      if (nullptr == out_coeffs) { return nof_coeffs; } // no allocated memory

      const bool is_valid_start_idx = start_idx < nof_coeffs;
      const bool is_valid_end_idx = end_idx < nof_coeffs && end_idx >= start_idx;
      const bool is_valid_indices = is_valid_start_idx && is_valid_end_idx;
      if (!is_valid_indices) {
        // return -1 instead? I could but 'get_coeff()' cannot with its current declaration
        THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "copy_coeffs() invalid indices");
      }

      op->transform_to_coefficients();
      auto [device_coeffs, _] = op->get_coefficients();
      const size_t nof_coeffs_to_copy = end_idx - start_idx + 1;
      ICICLE_CHECK(icicle_copy_async(out_coeffs, device_coeffs + start_idx, nof_coeffs_to_copy * sizeof(C), m_stream));
      ICICLE_CHECK(icicle_stream_synchronize(m_stream)); // sync to make sure return value is copied

      return nof_coeffs_to_copy;
    }

    // read coefficients to host
    C get_coeff(PolyContext op, uint64_t coeff_idx) override
    {
      C host_coeff;
      copy_coeffs(op, &host_coeff, coeff_idx, coeff_idx);
      return host_coeff;
    }

    std::tuple<IntegrityPointer<C>, uint64_t /*size*/> get_coefficients_view(PolyContext p) override
    {
      return p->get_coefficients_view();
    }

    inline void assert_device_compatability(const std::list<PolyContext>& polys) const
    {
      // TODO Yuval : move to context class
      for (const PolyContext& p : polys) {
        ICICLE_CHECK(icicle_is_active_device_memory(get_context_storage_immutable(p)));
      }
    }
  };

  /*============================== Polynomial CUDA-factory ==============================*/

#include "icicle/fields/field_config.h"

  template <typename C = scalar_t, typename D = C, typename I = C>
  class CUDAPolynomialFactory : public AbstractPolynomialFactory<C, D, I>
  {
    std::vector<hipStream_t> m_device_streams; // device-id --> device stream

  public:
    CUDAPolynomialFactory();
    ~CUDAPolynomialFactory();
    std::shared_ptr<IPolynomialContext<C, D, I>> create_context() override;
    std::shared_ptr<IPolynomialBackend<C, D, I>> create_backend() override;
  };

  template <typename C, typename D, typename I>
  CUDAPolynomialFactory<C, D, I>::CUDAPolynomialFactory()
  {
    int nof_cuda_devices = -1;
    CHK_STICKY(hipGetDeviceCount(&nof_cuda_devices));
    int orig_device = -1;

    CHK_STICKY(hipGetDevice(&orig_device));
    m_device_streams.resize(nof_cuda_devices, nullptr);

    for (int dev_id = 0; dev_id < nof_cuda_devices; ++dev_id) {
      CHK_STICKY(hipSetDevice(dev_id));
      CHK_STICKY(hipStreamCreate(&m_device_streams[dev_id]));
    }
    CHK_STICKY(hipSetDevice(orig_device)); // setting back original device
  }

  template <typename C, typename D, typename I>
  CUDAPolynomialFactory<C, D, I>::~CUDAPolynomialFactory()
  {
    for (auto stream_it : m_device_streams) {
      CHK_STICKY(hipStreamDestroy(stream_it));
    }
  }

  template <typename C, typename D, typename I>
  std::shared_ptr<IPolynomialContext<C, D, I>> CUDAPolynomialFactory<C, D, I>::create_context()
  {
    int cuda_device_id = -1;
    CHK_STICKY(hipGetDevice(&cuda_device_id));
    return std::make_shared<DefaultPolynomialContext<C, D, I>>(m_device_streams[cuda_device_id]);
  }

  template <typename C, typename D, typename I>
  std::shared_ptr<IPolynomialBackend<C, D, I>> CUDAPolynomialFactory<C, D, I>::create_backend()
  {
    int cuda_device_id = -1;
    CHK_STICKY(hipGetDevice(&cuda_device_id));
    return std::make_shared<DefaultPolynomialBackend<C, D, I>>(m_device_streams[cuda_device_id]);
  }

  /************************************** BACKEND REGISTRATION **************************************/

  REGISTER_SCALAR_POLYNOMIAL_FACTORY_BACKEND("CUDA", CUDAPolynomialFactory<scalar_t>)

} // namespace polynomials