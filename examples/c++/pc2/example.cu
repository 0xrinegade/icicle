#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <iostream>

// select the curve (only 2 available so far)
#define CURVE_ID 2
#include "appUtils/poseidon/poseidon.cu"
#include "appUtils/tree/merkle.cu"

using namespace poseidon;
using namespace merkle;
using namespace curve_config;

device_context::DeviceContext ctx= device_context::get_default_device_context();

void checkCudaError(hipError_t error) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        // Handle the error, e.g., exit the program or throw an exception.
    }
}

int main(int argc, char* argv[])
{
    // Outline of Filecoin PC2:
    // github.com/ingonyama-zk/research/blob/main/filecoin/doc/Filecoin.pdf

    std::cout << "Defining the size of the example" << std::endl;
    const uint32_t size_col=11;
    std::cout << "Number of layers: " << size_col << std::endl;
    const uint32_t height=9;
    const uint32_t height_icicle = height + 1;
    std::cout << "Tree height (edges, +1 to count levels): " << height <<  std::endl;
    const uint32_t tree_arity = 8;
    std::cout << "Tree arity: " << tree_arity << std::endl;
    const uint32_t size_row = pow(tree_arity,height); // (1<<(3*height));
    std::cout << "Tree width: " << size_row << std::endl;
 
    std::cout << "Allocating memory" << std::endl;
    scalar_t* layers = static_cast<scalar_t*>(malloc(size_col * size_row * sizeof(scalar_t)));
    scalar_t* column_hash = static_cast<scalar_t*>(malloc(size_row * sizeof(scalar_t)));


    std::cout << "Generating random inputs" << std::endl;
    scalar_t::RandHostMany(layers, size_col /* *size_row */);
    for (unsigned i = size_col; i < size_col*size_row; i++) {
        layers[i] = scalar_t::one();
    }
    std::cout << "Data generated" << std::endl;
    hipError_t err;
    std::cout << "Step 1: Column Hashing" << std::endl;
    PoseidonConstants<scalar_t> constants1;
    init_optimized_poseidon_constants<scalar_t>(size_col, ctx, &constants1);
    PoseidonConfig config1 = default_poseidon_config<scalar_t>(size_col+1);
    err = poseidon_hash<curve_config::scalar_t, size_col+1>(layers, column_hash, size_row, constants1, config1);
    checkCudaError(err);
    std::cout << "Step 2: Merkle Tree-C" << std::endl;
    auto digests_len = get_digests_len<scalar_t>(height_icicle, tree_arity);  // keep all digests
    // std::cout << "Digests length: " << digests_len << std::endl;
    scalar_t* digests = static_cast<scalar_t*>(malloc(digests_len * sizeof(scalar_t)));
    TreeBuilderConfig tree_config = default_merkle_config<scalar_t>(); // default: keep all digest raws
    PoseidonConstants<scalar_t> tree_constants;
    init_optimized_poseidon_constants<scalar_t>(tree_arity, ctx, &tree_constants);
    err = build_merkle_tree<scalar_t, tree_arity+1>(column_hash, digests, height_icicle, tree_constants, tree_config);
    checkCudaError(err);

    std::cout << "Cleaning up memory" << std::endl;
    free(layers);
    free(column_hash);
    free(digests);

    return 0;
}