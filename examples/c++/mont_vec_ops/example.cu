#include <iostream>
#include <iomanip>
#include <chrono>
#include <nvml.h>

#include "api/bn254.h"
#include "vec_ops/vec_ops.cuh"
#include <vec_ops/../../include/utils/mont.cuh>

using namespace vec_ops;
using namespace bn254;

typedef scalar_t T;

enum Op { MUL, ADD, SUB, LAST };

// bn254 p = 21888242871839275222246405745257275088548364400416034343698204186575808495617

int vector_op(
  T* vec_a,
  T* vec_b,
  T* vec_result,
  size_t n_elements,
  device_context::DeviceContext ctx,
  vec_ops::VecOpsConfig config,
  Op op)
{
  hipError_t err;
  switch (op) {
  case MUL:
    err = bn254_mul_cuda(vec_a, vec_b, n_elements, config, vec_result);
    break;
  case ADD:
    err = bn254_add_cuda(vec_a, vec_b, n_elements, config, vec_result);
    break;
  case SUB:
    err = bn254_sub_cuda(vec_a, vec_b, n_elements, config, vec_result);
    break;
  }
  // hipError_t err = bn254_mul_cuda(vec_a, vec_b, n_elements, config, vec_result);
  if (err != hipSuccess) {
    std::cerr << "Failed to multiply vectors - " << hipGetErrorString(err) << std::endl;
    return 0;
  }
  return 0;
}

int main(int argc, char** argv)
{
  const unsigned vector_size              = 1 << 11;
  const unsigned not_in_place_repetitions = 1 << 0;    // Repetitions are used only for the non in-place tests.
  const unsigned in_place_repetitions     = 1;    // Repetitions for in-place tests should be 1. Don't check it.

  hipError_t err;
  nvmlInit();
  nvmlDevice_t device;
  nvmlDeviceGetHandleByIndex(0, &device); // for GPU 0
  std::cout << "Icicle-Examples: vector mul / add / sub operations." << std::endl;
  char name[NVML_DEVICE_NAME_BUFFER_SIZE];
  if (nvmlDeviceGetName(device, name, NVML_DEVICE_NAME_BUFFER_SIZE) == NVML_SUCCESS) {
    std::cout << "GPU Model: " << name << std::endl;
  } else {
    std::cerr << "Failed to get GPU model name." << std::endl;
  }
  unsigned power_limit;
  nvmlDeviceGetPowerManagementLimit(device, &power_limit);

  std::cout << "Vector size:              " << vector_size << std::endl;
  std::cout << "not_in_place_repetitions: " << not_in_place_repetitions << std::endl;
  std::cout << "in_place_repetitions:     " << in_place_repetitions << std::endl;
  std::cout << "Power limit:              " << std::fixed << std::setprecision(3) << 1.0e-3 * power_limit << " W" << std::endl;

  unsigned int baseline_power;
  nvmlDeviceGetPowerUsage(device, &baseline_power);
  std::cout << "Baseline power: " << std::fixed << std::setprecision(3) << 1.0e-3 * baseline_power << " W" << std::endl;
  unsigned baseline_temperature;
  if (nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &baseline_temperature) == NVML_SUCCESS) {
    std::cout << "Baseline GPU Temperature: " << baseline_temperature << " C" << std::endl;
  } else {
    std::cerr << "Failed to get GPU temperature." << std::endl;
  }

  // host data
  std::cout << "Allocate memory for the input vectors (both normal and Montgomery presentation)" << std::endl;
  T* host_in1_init = (T*)malloc(vector_size * sizeof(T));
  std::cout << "example malloc host_in1_init" << std::endl;
  T* host_in2_init = (T*)malloc(vector_size * sizeof(T));
  std::cout << "example malloc host_in2_init" << std::endl;
  std::cout << "Initializing vectors with normal presentation random data" << std::endl;
  T::rand_host_many(host_in1_init, vector_size);
  T::rand_host_many(host_in2_init, vector_size);
  std::cout << "Allocate memory for the output vectors" << std::endl;
  T* host_out = (T*)malloc(vector_size * sizeof(T)); // This memory will be used for the test output.
  std::cout << "example malloc host_out" << std::endl;
  T* host_out_ref_mul = (T*)malloc(
    vector_size *
    sizeof(T)); // This memory will be used as a reference result for mul (will be compared to host_out content).
  std::cout << "example malloc host_out_ref_mul" << std::endl;
  T* host_out_ref_add = (T*)malloc(
    vector_size *
    sizeof(T)); // This memory will be used as a reference result for add (will be compared to host_out content).
  std::cout << "example malloc host_out_ref_add" << std::endl;    
  T* host_out_ref_sub = (T*)malloc(
    vector_size *
    sizeof(T)); // This memory will be used as a reference result for sub (will be compared to host_out content).
  std::cout << "malloc host_out_ref_sub" << std::endl;
  std::cout << "Initializing output vectors with random data" << std::endl;
  T::rand_host_many(host_out, vector_size);
  T::rand_host_many(host_out_ref_mul, vector_size);
  T::rand_host_many(host_out_ref_add, vector_size);
  T::rand_host_many(host_out_ref_sub, vector_size);
  // device data
  device_context::DeviceContext ctx = device_context::get_default_device_context();
  T* device_in1;
  T* device_in2;
  T* device_out;

  err = hipMalloc((void**)&device_in1, vector_size * sizeof(T));
  std::cout << "example hipMalloc device_in1" << std::endl;
  if (err != hipSuccess) {
    std::cerr << "Failed to allocate device memory - " << hipGetErrorString(err) << std::endl;
    return 0;
  }
  err = hipMalloc((void**)&device_in2, vector_size * sizeof(T));
  std::cout << "example hipMalloc device_in2" << std::endl;
  if (err != hipSuccess) {
    std::cerr << "Failed to allocate device memory - " << hipGetErrorString(err) << std::endl;
    return 0;
  }
  err = hipMalloc((void**)&device_out, vector_size * sizeof(T));
  std::cout << "example hipMalloc device_out" << std::endl;
  if (err != hipSuccess) {
    std::cerr << "Failed to allocate device memory - " << hipGetErrorString(err) << std::endl;
    return 0;
  }

  vec_ops::VecOpsConfig config = vec_ops::DefaultVecOpsConfig();
  int nof_of_configs_for_test = 5;
  int nof_of_storage_configs = 3;   // 2 inputs, 1 result.

  //****************************************
  // Test warn-up and reference output config. Reference output to be used to check if test passed or not.
  //****************************************
  // copy from host to device
  err = hipMemcpy(device_in1, host_in1_init, vector_size * sizeof(T), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    std::cerr << "Failed to copy data from host to device - " << hipGetErrorString(err) << std::endl;
    return 0;
  }
  err = hipMemcpy(device_in2, host_in2_init, vector_size * sizeof(T), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    std::cerr << "Failed to copy data from host to device - " << hipGetErrorString(err) << std::endl;
    return 0;
  }
  std::cout << "Starting warm-up run" << std::endl;
  // Warm-up loop
  for (int op = MUL; op != LAST; op++) {
    for (int i = 0; i < not_in_place_repetitions; i++) {
      vector_op(device_in1, device_in2, device_out, vector_size, ctx, config, (Op)op);
      switch (op) {
        case MUL:
          err = hipMemcpy(host_out_ref_mul, device_out, vector_size * sizeof(T), hipMemcpyDeviceToHost);
          break;
        case ADD:
          err = hipMemcpy(host_out_ref_add, device_out, vector_size * sizeof(T), hipMemcpyDeviceToHost);
          break;
        case SUB:
          err = hipMemcpy(host_out_ref_sub, device_out, vector_size * sizeof(T), hipMemcpyDeviceToHost);
          break;
      }
      if (err != hipSuccess) {
        std::cerr << "Failed to copy data from device_out to host - " << hipGetErrorString(err) << std::endl;
        return 0;
      }
    }
  }
  //****************************************
  // End of test warn-up and reference output config.
  //****************************************

  std::cout << "Starting benchmarking" << std::endl;
  unsigned power_before;
  nvmlDeviceGetPowerUsage(device, &power_before);
  std::cout << "Power before: " << std::fixed << std::setprecision(3) << 1.0e-3 * power_before << " W" << std::endl;
  std::cout << "Power utilization: " << std::fixed << std::setprecision(1) << (float)100.0 * power_before / power_limit
            << " %" << std::endl;
  unsigned temperature_before;
  if (nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature_before) == NVML_SUCCESS) {
    std::cout << "GPU Temperature before: " << temperature_before << " C" << std::endl;
  } else {
    std::cerr << "Failed to get GPU temperature." << std::endl;
  }

  //*******************************************************
  // Benchmark test:
  // Loop for (mul, add, sub):
  //   Loop (is_a_on_device, is_b_on_device, is_result_on_device, is_input_in_montgomery_form):
  //*******************************************************
  T* host_in1 =
    (T*)malloc(vector_size * sizeof(T)); // This buffer is used to load the data from host_in1_init for the benchmark.
  std::cout << "example malloc host_in1" << std::endl;
  T* host_in2 =
    (T*)malloc(vector_size * sizeof(T)); // This buffer is used to load the data from host_in2_init for the benchmark.
  std::cout << "example malloc host_in1" << std::endl;
  // Test when the result is not in-place
  std::cout << "*****************************************" << std::endl;
  std::cout << "*** Start not in-place benchmark loop ***" << std::endl;
  std::cout << "*****************************************" << std::endl;
  for (int op = MUL; op != LAST; op++) {
    for (int config_idx = 0; config_idx < 32; config_idx++) {
      // // DEBUG
      // int num_gpus;
      // size_t free, total;
      // hipGetDeviceCount( &num_gpus );
      // std::cout << "num_gpus " << num_gpus << std::endl;
      // for ( int gpu_id = 0; gpu_id < num_gpus; gpu_id++ ) {
      //     hipSetDevice( gpu_id );
      //     int id;
      //     hipGetDevice( &id );
      //     hipMemGetInfo( &free, &total );
      //     std::cout << "GPU " << id << " memory: free=" << free << ", total=" << total << std::endl;
      // }
      // // DEBUG
      switch (op) {
        case MUL: std::cout << "Start benchmark loop for op MUL config_idx " << config_idx << " not in-place" << std::endl; break;
        case ADD: std::cout << "Start benchmark loop for op ADD config_idx " << config_idx << " not in-place" << std::endl; break;
        case SUB: std::cout << "Start benchmark loop for op SUB config_idx " << config_idx << " not in-place" << std::endl; break;
      }
      std::cout << "Start benchmark loop for config_idx " << config_idx << std::endl;
      // Destroy the result of the prev loop.
      T::rand_host_many(host_out, vector_size);   // Randomize host_out in order to randomize device_out.
      err = hipMemcpy(
        device_out, host_out, vector_size * sizeof(T),
        hipMemcpyHostToDevice); // Copy random data to device_out.
      if (err != hipSuccess) {
        std::cerr << "Failed to copy data from host_out to device_out - " << hipGetErrorString(err) << std::endl;
        return 0;
      }
      T::rand_host_many(host_out, vector_size);   // Make hist_out != device_out. 
      // Initializa inputs with the known data
      for (int i = 0; i < vector_size; i++) {
        host_in1[i] = host_in1_init[i];
        host_in2[i] = host_in2_init[i];
      }
      config.is_a_on_device               = (config_idx >> 4) & 0x1;
      config.is_b_on_device               = (config_idx >> 3) & 0x1;
      config.is_result_on_device          = (config_idx >> 2) & 0x1;
      config.is_input_in_montgomery_form  = (config_idx >> 1) & 0x1;
      config.is_result_in_montgomery_form = (config_idx >> 0) & 0x1;

      // Copy from host to device (copy again in order to be used later in the loop and device_inX was already
      // overwritten by warmup.
      if (config.is_a_on_device) {
        if (config.is_input_in_montgomery_form) {
          err =
            hipMemcpy(device_in1, host_in1, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in1 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(
            mont::to_montgomery(device_in1, vector_size, config.ctx.stream, device_in1)); // Convert in-place.
          // Destroy host_in1 value with values of host_in2.
          for (int i = 0; i < vector_size; i++) {
            host_in1[i] = host_in2_init[i];
          }
        } else {                                                                          // Normal presentation.
          err =
            hipMemcpy(device_in1, host_in1, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in1 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        }
      } else {
        if (config.is_input_in_montgomery_form) { // Copy to device, cnvert to montgomery and copy back to host.
          err =
            hipMemcpy(device_in1, host_in1, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in1 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(mont::to_montgomery(device_in1, vector_size, config.ctx.stream, device_in1));
          err = hipMemcpy(host_in1, device_in1, vector_size * sizeof(T), hipMemcpyDeviceToHost);
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from device_in1 to host_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          // Destroy device_in1 value with values of host_in2.
          err =
            hipMemcpy(device_in1, host_in2, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in2 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        }
      }
      if (config.is_b_on_device) {
        if (config.is_input_in_montgomery_form) {
          err =
            hipMemcpy(device_in2, host_in2, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in2 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(
            mont::to_montgomery(device_in2, vector_size, config.ctx.stream, device_in2)); // Convert in-place.
          // Destroy host_in2 value with values of host_in1.
          for (int i = 0; i < vector_size; i++) {
            host_in2[i] = host_in1_init[i];
          }
        } else {
          // Normal presentation.
          err =
            hipMemcpy(device_in2, host_in2, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in2 to device_in2 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        }
      } else {
        if (config.is_input_in_montgomery_form) { // Copy to device, cnvert to montgomery and copy back to host.
          err =
            hipMemcpy(device_in2, host_in2, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in2 to device_in2 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(mont::to_montgomery(device_in2, vector_size, config.ctx.stream, device_in2));
          err = hipMemcpy(host_in2, device_in2, vector_size * sizeof(T), hipMemcpyDeviceToHost);
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from device_in2 to host_in2 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          // Destroy device_in2 valuewith values of host_in1.
          err =
            hipMemcpy(device_in2, host_in1, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in1 to device_in2 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        }
      }
      CHK_IF_RETURN(hipPeekAtLastError());

      auto start_time = std::chrono::high_resolution_clock::now();
      // Benchmark loop
      for (int i = 0; i < not_in_place_repetitions; i++) {
        switch (config_idx >> (nof_of_configs_for_test - nof_of_storage_configs)) { // {is_a_on_device, is_b_on_device, is_result_on_device}
          case 0b000:
            vector_op(host_in1, host_in2, host_out, vector_size, ctx, config, (Op)op);
            break;
          case 0b001:
            vector_op(host_in1, host_in2, device_out, vector_size, ctx, config, (Op)op);
            break;
          case 0b010:
            vector_op(host_in1, device_in2, host_out, vector_size, ctx, config, (Op)op);
            break;
          case 0b011:
            vector_op(host_in1, device_in2, device_out, vector_size, ctx, config, (Op)op);
            break;
          case 0b100:
            vector_op(device_in1, host_in2, host_out, vector_size, ctx, config, (Op)op);
            break;
          case 0b101:
            vector_op(device_in1, host_in2, device_out, vector_size, ctx, config, (Op)op);
            break;
          case 0b110:
            vector_op(device_in1, device_in2, host_out, vector_size, ctx, config, (Op)op);
            break;
          case 0b111:
            vector_op(device_in1, device_in2, device_out, vector_size, ctx, config, (Op)op);
            break;
        }
        CHK_IF_RETURN(hipPeekAtLastError());
      }

      auto end_time = std::chrono::high_resolution_clock::now();
      auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
      switch (op) {
      case MUL:
        std::cout << "Elapsed time: " << duration.count() << " microseconds, operation MUL for config_idx "
                  << config_idx << " and result not in-place" << std::endl;
        break;
      case ADD:
        std::cout << "Elapsed time: " << duration.count() << " microseconds, operation ADD for config_idx "
                  << config_idx << " and result not in-place" << std::endl;
        break;
      case SUB:
        std::cout << "Elapsed time: " << duration.count() << " microseconds, operation SUB for config_idx "
                  << config_idx << " and result not in-place" << std::endl;
        break;
      }

      if (config.is_result_on_device) { // Copy the data to host_out in order to compare it vs. host_out_ref_XXX value.
        if (config.is_result_in_montgomery_form) { // Convert to normal from montgomery if needed.
          CHK_IF_RETURN(mont::from_montgomery(
            device_out, vector_size, config.ctx.stream,
            device_out)); // Convert to normal in order to check vs. host_out_ref_XXX.
        }
        err = hipMemcpy(
          host_out, device_out, vector_size * sizeof(T),
          hipMemcpyDeviceToHost); // Copy to host_out in order to check vs. host_out_ref_XXX.
        if (err != hipSuccess) {
          std::cerr << "Failed to copy data from device_out to host - " << hipGetErrorString(err) << std::endl;
          return 0;
        }
      } else {                              // Data is not on device but it is in host_out.
        if (config.is_result_in_montgomery_form) { // host_out should be written to device, converted to mmontgomery and
                                                   // written back to host. Then compared vs. host_out_ref_XXX.
          err = hipMemcpy(
            device_out, host_out, vector_size * sizeof(T),
            hipMemcpyHostToDevice); // Copy to device_out in order to check later vs. host_out_ref_XXX.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_out to device_out - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(mont::from_montgomery(
            device_out, vector_size, config.ctx.stream,
            device_out)); // Convert to normal in order to check vs. host_out_ref_XXX.
          err = hipMemcpy(
            host_out, device_out, vector_size * sizeof(T),
            hipMemcpyDeviceToHost); // Copy to host_out in order to check vs. host_out_ref_XXX.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from device_out to host_out - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        } else { // host_out could be compared vs. host_out_ref_XXX as is.   
        }
      }
      //****************************************
      // End of benchmark test.
      //****************************************

      //***********************************************
      // Test result check (not in-place)
      // Check is performed by executing the operation in a normal presentation
      //   (located in in host_out_ref_XXX) and comparing it with the
      //   benchmark test result.
      //***********************************************
      int test_failed = 0;
      switch (op) {
      case MUL:
        for (int i = 0; i < vector_size; i++) {
          if (host_out_ref_mul[i] != host_out[i]) {
            std::cout << "===>>> ERROR!!! MUL: Test failed for vector index " << i
                      << ", config is printed below:" << std::endl;
            test_failed = 1;
          }
        }
        break;
      case ADD:
        for (int i = 0; i < vector_size; i++) {
          if (host_out_ref_add[i] != host_out[i]) {
            std::cout << "===>>> ERROR!!! ADD: Test failed for vector index " << i
                      << ", config is printed below:" << std::endl;
            test_failed = 1;
          }
        }
        break;
      case SUB:
        for (int i = 0; i < vector_size; i++) {
          if (host_out_ref_sub[i] != host_out[i]) {
            std::cout << "===>>> ERROR!!! SUB: Test failed for vector index " << i
                      << ", config is printed below:" << std::endl;
            test_failed = 1;
          }
        }
        break;
      }
      if (test_failed) {
        std::cout << "===>>> result is in-place:                " << std::endl;
        std::cout << "===>>> is_a_on_device:                    " << config.is_a_on_device << std::endl;
        std::cout << "===>>> is_b_on_device:                    " << config.is_b_on_device << std::endl;
        std::cout << "===>>> is_result_on_device:               " << config.is_result_on_device << std::endl;
        std::cout << "===>>> is_input_in_montgomery_form:       " << config.is_input_in_montgomery_form << std::endl;
        std::cout << "===>>> is_input_resultin_montgomery_form: " << config.is_result_in_montgomery_form << std::endl;
        std::cout << "===>>> host_in1_init[0]                      = " << host_in1_init[0] << std::endl;
        std::cout << "===>>> host_in2_init[0]                      = " << host_in2_init[0] << std::endl;
        std::cout << "===>>> host_out[0]                           = " << host_out[0] << std::endl;
        std::cout << "===>>> warm-up: normal host_out_ref[0] (MUL) = " << host_out_ref_mul[0] << std::endl;
        std::cout << "===>>> warm-up: normal host_out_ref[0] (ADD) = " << host_out_ref_add[0] << std::endl;
        std::cout << "===>>> warm-up: normal host_out_ref[0] (SUB) = " << host_out_ref_sub[0] << std::endl;
        exit(2);
      }

      unsigned power_after;
      nvmlDeviceGetPowerUsage(device, &power_after);
      std::cout << "Power after: " << std::fixed << std::setprecision(3) << 1.0e-3 * power_after << " W" << std::endl;
      std::cout << "Power utilization: " << std::fixed << std::setprecision(1)
                << (float)100.0 * power_after / power_limit << " %" << std::endl;
      unsigned temperature_after;
      if (nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature_after) == NVML_SUCCESS) {
        std::cout << "GPU Temperature after: " << temperature_after << " C" << std::endl;
      } else {
        std::cerr << "Failed to get GPU temperature." << std::endl;
      }

      // Report performance in GMPS: Giga Multiplications Per Second
      double GMPS = 1.0e-9 * not_in_place_repetitions * vector_size / (1.0e-6 * duration.count());
      std::cout << "Performance: " << GMPS << " Giga Multiplications Per Second" << std::endl;
    }
  }

  // Test when the result is in-place
  std::cout << "*************************************" << std::endl;
  std::cout << "*** Start in-place benchmark loop ***" << std::endl;
  std::cout << "*************************************" << std::endl;
  for (int op = MUL; op != LAST; op++) {
    for (int config_idx = 0; config_idx < 32; config_idx++) {
      switch (op) {
        case MUL: std::cout << "Start benchmark loop for op MUL config_idx " << config_idx << " in-place" << std::endl; break;
        case ADD: std::cout << "Start benchmark loop for op ADD config_idx " << config_idx << " in-place" << std::endl; break;
        case SUB: std::cout << "Start benchmark loop for op SUB config_idx " << config_idx << " in-place" << std::endl; break;
      }
      std::cout << "Start benchmark loop for config_idx " << config_idx << std::endl;
      // Destroy the result of the prev loop.
      T::rand_host_many(host_out, vector_size);   // Randomize host_out in order to randomize device_out.
      err = hipMemcpy(
        device_out, host_out, vector_size * sizeof(T),
        hipMemcpyHostToDevice); // Copy random data to device_out.
      if (err != hipSuccess) {
        std::cerr << "Failed to copy data from host_out to device_out - " << hipGetErrorString(err) << std::endl;
        return 0;
      }
      T::rand_host_many(host_out, vector_size);   // Make hist_out != device_out. 
      // Initializa inputs with the known data. For in-place tests host_in1 is going to be used as a result. So, it should be initialized later in the repetions loop.
      for (int i = 0; i < vector_size; i++) {
        host_in1[i] = host_in1_init[i];
        host_in2[i] = host_in2_init[i];
      }
      config.is_a_on_device               = (config_idx >> 4) & 0x1;
      config.is_b_on_device               = (config_idx >> 3) & 0x1;
      config.is_result_on_device          = (config_idx >> 2) & 0x1;
      config.is_input_in_montgomery_form  = (config_idx >> 1) & 0x1;
      config.is_result_in_montgomery_form = (config_idx >> 1) & 0x1;
      if (config.is_a_on_device ^ config.is_result_on_device == 1) { continue; }    // Illegal case for this loop.
      if (config.is_input_in_montgomery_form ^ config.is_result_in_montgomery_form == 1) { continue; }    // Illegal case for this loop.

      // Copy from host to device (copy again in order to be used later in the loop and device_inX was already
      // overwritten by warmup.
      if (config.is_a_on_device) {
        if (config.is_input_in_montgomery_form) {
          err =
            hipMemcpy(device_in1, host_in1, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in1 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(
            mont::to_montgomery(device_in1, vector_size, config.ctx.stream, device_in1)); // Convert in-place.
          // Destroy host_in1 value with values of host_in2.
          for (int i = 0; i < vector_size; i++) {
            host_in1[i] = host_in2_init[i];
          }
        } else {                                                                          // Normal presentation.
          err =
            hipMemcpy(device_in1, host_in1, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in1 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        }
      } else {
        if (config.is_input_in_montgomery_form) { // Copy to device, cnvert to montgomery and copy back to host.
          err =
            hipMemcpy(device_in1, host_in1, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in1 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(mont::to_montgomery(device_in1, vector_size, config.ctx.stream, device_in1));
          err = hipMemcpy(host_in1, device_in1, vector_size * sizeof(T), hipMemcpyDeviceToHost);
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from device_in1 to host_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          // Destroy device_in1 value with values of host_in2.
          err =
            hipMemcpy(device_in1, host_in2, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in2 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        }
      }
      if (config.is_b_on_device) {
        if (config.is_input_in_montgomery_form) {
          err =
            hipMemcpy(device_in2, host_in2, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in2 to device_in1 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(
            mont::to_montgomery(device_in2, vector_size, config.ctx.stream, device_in2)); // Convert in-place.
          // Destroy host_in2 value with values of host_in1.
          for (int i = 0; i < vector_size; i++) {
            host_in2[i] = host_in1_init[i];
          }
        } else {
          // Normal presentation.
          err =
            hipMemcpy(device_in2, host_in2, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in2 to device_in2 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        }
      } else {
        if (config.is_input_in_montgomery_form) { // Copy to device, cnvert to montgomery and copy back to host.
          err =
            hipMemcpy(device_in2, host_in2, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in2 to device_in2 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(mont::to_montgomery(device_in2, vector_size, config.ctx.stream, device_in2));
          err = hipMemcpy(host_in2, device_in2, vector_size * sizeof(T), hipMemcpyDeviceToHost);
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from device_in2 to host_in2 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          // Destroy device_in2 valuewith values of host_in1.
          err =
            hipMemcpy(device_in2, host_in1, vector_size * sizeof(T), hipMemcpyHostToDevice); // Copy data to device.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in1 to device_in2 - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        }
      }
      CHK_IF_RETURN(hipPeekAtLastError());

      auto start_time = std::chrono::high_resolution_clock::now();
      // Benchmark loop
      for (int i = 0; i < in_place_repetitions; i++) {
        switch (config_idx >> (nof_of_configs_for_test - nof_of_storage_configs)) { // {is_a_on_device, is_b_on_device, is_result_on_device}
          case 0b000:
            vector_op(host_in1, host_in2, host_in1, vector_size, ctx, config, (Op)op);
            break;
          case 0b001:
            break;
          case 0b010:
            vector_op(host_in1, device_in2, host_in1, vector_size, ctx, config, (Op)op);
            break;
          case 0b011:
            break;
          case 0b100:
            break;
          case 0b101:
            vector_op(device_in1, host_in2, device_in1, vector_size, ctx, config, (Op)op);
            std::cout << "===>>> COMMAND: vector_op(device_in1, host_in2, device_in1, vector_size, ctx, config, (Op)op);" << std::endl;
            break;
          case 0b110:
            break;
          case 0b111:
            vector_op(device_in1, device_in2, device_in1, vector_size, ctx, config, (Op)op);
            break;
        }
        CHK_IF_RETURN(hipPeekAtLastError());
      }

      auto end_time = std::chrono::high_resolution_clock::now();
      auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
      switch (op) {
      case MUL:
        std::cout << "Elapsed time: " << duration.count() << " microseconds, operation MUL for config_idx "
                  << config_idx << " and result in-place" << std::endl;
        break;
      case ADD:
        std::cout << "Elapsed time: " << duration.count() << " microseconds, operation ADD for config_idx "
                  << config_idx << " and result in-place" << std::endl;
        break;
      case SUB:
        std::cout << "Elapsed time: " << duration.count() << " microseconds, operation SUB for config_idx "
                  << config_idx << " and result in-place" << std::endl;
        break;
      }

      if (config.is_result_on_device) { // Copy the data to host_out in order to compare it vs. host_out_ref_XXX value.
        if (config.is_result_in_montgomery_form) { // Convert to normal from montgomery if needed.
          CHK_IF_RETURN(mont::from_montgomery(
            device_in1, vector_size, config.ctx.stream,
            device_in1)); // Convert to normal in order to check vs. host_out_ref_XXX.
        }
        err = hipMemcpy(
          host_out, device_in1, vector_size * sizeof(T),
          hipMemcpyDeviceToHost); // Copy to host_out in order to check vs. host_out_ref_XXX.
        if (err != hipSuccess) {
          std::cerr << "Failed to copy data from device_in1 to host_out - " << hipGetErrorString(err) << std::endl;
          return 0;
        }
      } else { // Data is not on device but it is in host_in1. It should be moved to host_out for test pass/fail check.
        if (config.is_result_in_montgomery_form) { // host_out should be written to device, converted to mmontgomery and
                                                   // written back to host. Then compared vs. host_out_ref_XXX.
          err = hipMemcpy(
            device_out, host_in1, vector_size * sizeof(T),
            hipMemcpyHostToDevice); // Copy to device_out in order to check later vs. host_out_ref_XXX.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from host_in1 to device_out - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
          CHK_IF_RETURN(mont::from_montgomery(
            device_out, vector_size, config.ctx.stream,
            device_out)); // Convert to normal in order to check vs. host_out_ref_XXX.
          err = hipMemcpy(
            host_out, device_out, vector_size * sizeof(T),
            hipMemcpyDeviceToHost); // Copy to host_out in order to check vs. host_out_ref_XXX.
          if (err != hipSuccess) {
            std::cerr << "Failed to copy data from device_out to host_out - " << hipGetErrorString(err) << std::endl;
            return 0;
          }
        } else { // host_out could be compared vs. host_out_ref_XXX as is.
          for (int i = 0; i < vector_size; i++) {   // Copy from host_in1 (result) to host_out to compare later vs. host_out_ref_XXX.
            host_out[i] = host_in1[i];
          }
        }
      }
      //****************************************
      // End of benchmark test.
      //****************************************

      //***********************************************
      // Test result check (in-place)
      // Check is performed by executing the operation in a normal presentation
      //   (located in in host_out_ref_XXX) and comparing it with the
      //   benchmark test result.
      //***********************************************
      int test_failed = 0;
      switch (op) {
        case MUL:
          for (int i = 0; i < vector_size; i++) {
            if (host_out_ref_mul[i] != host_out[i]) {
              std::cout << "===>>> ERROR!!! MUL: Test failed for vector index " << i
                        << ", config is printed below:" << std::endl;
              test_failed = 1;
            }
          }
          break;
        case ADD:
          for (int i = 0; i < vector_size; i++) {
            if (host_out_ref_add[i] != host_out[i]) {
              std::cout << "===>>> ERROR!!! ADD: Test failed for vector index " << i
                        << ", config is printed below:" << std::endl;
              test_failed = 1;
            }
          }
          break;
        case SUB:
          for (int i = 0; i < vector_size; i++) {
            if (host_out_ref_sub[i] != host_out[i]) {
              std::cout << "===>>> ERROR!!! SUB: Test failed for vector index " << i
                        << ", config is printed below:" << std::endl;
              test_failed = 1;
            }
          }
          break;
      }
      if (test_failed) {
        std::cout << "===>>> result is in-place:                " << std::endl;
        std::cout << "===>>> is_a_on_device:                    " << config.is_a_on_device << std::endl;
        std::cout << "===>>> is_b_on_device:                    " << config.is_b_on_device << std::endl;
        std::cout << "===>>> is_result_on_device:               " << config.is_result_on_device << std::endl;
        std::cout << "===>>> is_input_in_montgomery_form:       " << config.is_input_in_montgomery_form << std::endl;
        std::cout << "===>>> is_input_resultin_montgomery_form: " << config.is_result_in_montgomery_form << std::endl;
        std::cout << "===>>> host_in1_init[0]                      = " << host_in1_init[0] << std::endl;
        std::cout << "===>>> host_in2_init[0]                      = " << host_in2_init[0] << std::endl;
        std::cout << "===>>> host_out[0]                           = " << host_out[0] << std::endl;
        std::cout << "===>>> warm-up: normal host_out_ref[0] (MUL) = " << host_out_ref_mul[0] << std::endl;
        std::cout << "===>>> warm-up: normal host_out_ref[0] (ADD) = " << host_out_ref_add[0] << std::endl;
        std::cout << "===>>> warm-up: normal host_out_ref[0] (SUB) = " << host_out_ref_sub[0] << std::endl;
        exit(2);
      }

      unsigned power_after;
      nvmlDeviceGetPowerUsage(device, &power_after);
      std::cout << "Power after: " << std::fixed << std::setprecision(3) << 1.0e-3 * power_after << " W" << std::endl;
      std::cout << "Power utilization: " << std::fixed << std::setprecision(1)
                << (float)100.0 * power_after / power_limit << " %" << std::endl;
      unsigned temperature_after;
      if (nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature_after) == NVML_SUCCESS) {
        std::cout << "GPU Temperature after: " << temperature_after << " C" << std::endl;
      } else {
        std::cerr << "Failed to get GPU temperature." << std::endl;
      }

      // Report performance in GMPS: Giga Multiplications Per Second
      double GMPS = 1.0e-9 * in_place_repetitions * vector_size / (1.0e-6 * duration.count());
      std::cout << "Performance: " << GMPS << " Giga Multiplications Per Second" << std::endl;
    }
  }

  // clean up and exit
  free(host_in1_init);
  std::cout << "example free host_in1_init" << std::endl;
  free(host_in2_init);
  std::cout << "example free host_in2_init" << std::endl;
  free(host_in1);
  std::cout << "example free host_in1" << std::endl;
  free(host_in2);
  std::cout << "example free host_in2" << std::endl;
  free(host_out);
  std::cout << "example free host_out" << std::endl;
  free(host_out_ref_mul);
  std::cout << "example free host_out_ref_mul" << std::endl;
  free(host_out_ref_add);
  std::cout << "example free host_out_ref_add" << std::endl;
  free(host_out_ref_sub);
  std::cout << "example free host_out_ref_sub" << std::endl;
  hipFree(device_in1);
  std::cout << "example hipFree device_in1" << std::endl;
  hipFree(device_in2);
  std::cout << "example hipFree device_in2" << std::endl;
  hipFree(device_out);
  std::cout << "example hipFree device_out" << std::endl;
  nvmlShutdown();
  return 0;
}
